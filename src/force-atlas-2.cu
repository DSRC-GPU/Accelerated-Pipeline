#include "hip/hip_runtime.h"
/*!
 * \file force-atlas-2.cu
 * A parallel implementation of the Force Atlas 2 spring embedding algorithm.
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <float.h>
#include "force-atlas-2.h"
#include "math.h"
#include "cuda-timer.h"
#include "cuda-stream.h"
#include "vector.h"
#include "speedvector.h"
#include "util.h"

/*!
 * Updates the current force on each vertex with the current gravity.
 *
 * \param[in] gid The global ID of this thread.
 * \param[in] numvertices The total number of vertices in the graph.
 * \param[in] vxLocs Array that holds the x location of all vertices.
 * \param[in] vyLocs Array that holds the y location of all vertices.
 * \param[out] forceX Pointer to the x force on the vertex that belongs to this
 *    thread.
 * \param[out] forceY Pointer to the y force on the vertex that belongs to this
 *    thread.
 * \param[in] deg Array holding the out degree values for each vertex.
 */
__device__ void fa2Gravity(unsigned int gid, unsigned int numvertices,
    float* vxLocs, float* vyLocs, float* forceX, float* forceY,
    unsigned int* deg);

/*!
 * Updates the current force on each vertex with the current repulsion.
 *
 * \param[in] gid The global ID of this thread.
 * \param[in] numvertices The total number of vertices in the graph.
 * \param[in] vxLocs Array that holds the x location of all vertices.
 * \param[in] vyLocs Array that holds the y location of all vertices.
 * \param[out] forceX Pointer to the x force on the vertex that belongs to this
 *    thread.
 * \param[out] forceY Pointer to the y force on the vertex that belongs to this
 *    thread.
 * \param[in] deg Array holding the out degree values for each vertex.
 */
__device__ void fa2Repulsion(unsigned int gid, unsigned int numvertices,
    float* vxLocs, float* vyLocs, float* forceX, float* forceY,
    unsigned int* deg);

/*!
 * Updates the current force on each vertex with the current attraction.
 *
 * \param[in] gid The global ID of this thread.
 * \param[in] numvertices The total number of vertices in the graph.
 * \param[in] vxLocs Array that holds the x location of all vertices.
 * \param[in] vyLocs Array that holds the y location of all vertices.
 * \param[in] numedges The total number of edges in the graph.
 * \param[in] edgeTargets Array holding the edge targets for all edges.
 * \param[in] maxedges The maximum number of edges per vertex.
 * \param[out] forceX Pointer to the x force on the vertex that belongs to this
 *    thread.
 * \param[out] forceY Pointer to the y force on the vertex that belongs to this
 *    thread.
 */
__device__ void fa2Attraction(unsigned int gid, unsigned int numvertices,
    float* vxLocs, float* vyLocs, unsigned int* numedges,
    unsigned int* edgeTargets, unsigned int maxedges, float* forceX,
    float* forceY);

/*!
 * Updates the swing value for each vertex in the graph.
 *
 * \param[in] gid The global ID of this thread.
 * \param[in] numvertices The total number of vertices in the graph.
 * \param[in] forceX The x force value on the vertex that belongs to this
 *    thread.
 * \param[in] forceY The y force value on the vertex that belongs to this
 *    thread.
 * \param[in] oldForceX Array holding the x forces of each vertex from the
 *    previous iteration.
 * \param[in] oldForceY Array holding the y forces of each vertex from the
 *    previous iteration.
 * \param[out] swg Array where the swing values for each vertex should be
 *    stored.
 */
__device__ void fa2UpdateSwing(unsigned int gid, unsigned int numvertices,
    float forceX, float forceY, float* oldForceX, float* oldForceY, float* swg);

/*!
 * Updates the traction value for each vertex in the graph.
 *
 * \param[in] gid The global ID of this thread.
 * \param[in] numvertices The total number of vertices in the graph.
 * \param[in] forceX The x force value on the vertex that belongs to this
 *    thread.
 * \param[in] forceY The y force value on the vertex that belongs to this
 *    thread.
 * \param[in] oldForceX Array holding the x forces of each vertex from the
 *    previous iteration.
 * \param[in] oldForceY Array holding the y forces of each vertex from the
 *    previous iteration.
 * \param[out] tra Array where the traction values for each vertex should be
 *    stored.
 */
__device__ void fa2UpdateTract(unsigned int gid, unsigned int numvertices,
    float forceX, float forceY, float* oldForceX, float* oldForceY, float* tra);

/*!
 * Updates the speed value for the graph itself.
 *
 * \param[in] gswing The swing value of the graph.
 * \param[in] gtract The traction value of the graph.
 * \param[out] gspeed Pointer where the graph speed should be stored.
 */
__device__ void fa2UpdateSpeedGraph(float gswing, float gtract, float* gspeed);

/*!
 * Updates the speed value for each vertex in the graph.
 *
 * \param[in] gid The global ID of this thread.
 * \param[in] numvertices The total number of vertices in the graph.
 * \param[out] speed Pointer to where the speed value of the vertex that belongs
 *    to this thread should be stored.
 * \param[in] swg An array holding the swing values for each vertex.
 * \param[in] forceX The x force on the vertex that belongs to this thread.
 * \param[in] forceY The y force on the vertex that belongs to this thread.
 * \param[in] gs The graph speed value.
 */
__device__ void fa2UpdateSpeed(unsigned int gid, unsigned int numvertices,
    float* speed, float* swg, float forceX, float forceY, float gs);

/*!
 * Copies the forces of this iteration to another array. Overwrites the values
 * in the destination array.
 *
 * \param[in] gid The global ID of this thread.
 * \param[in] numvertices The total number of vertices in the graph.
 * \param[in] forceX The x force on the vertex that belongs to this thread.
 * \param[in] forceY The y force on the vertex that belongs to this thread.
 * \param[out] oldForceX Array that will be used to store the x forces of this
 *    iteration.
 * \param[out] oldForceY Array that will be used to store the y forces of this
 *    iteration.
 */
__device__ void fa2SaveOldForces(unsigned int gid, unsigned int numvertices,
    float forceX, float forceY, float* oldForceX, float* oldForceY);

/*!
 * Updates the vertex displacement array.
 *
 * \param[in] gid The global ID of this thread.
 * \param[in] numvertices The total number of vertices in the graph.
 * \param[in] speed The speed value for the vertex that belongs to this thread.
 * \param[in] forceX The x force value for the vertex that belongs to this
 *    thread.
 * \param[in] forceY The y force value for the vertex that belongs to this
 *    thread.
 * \param[out] dispX Pointer to where the x displacement for the vertex that
 *    belongs to this thread should be stored.
 * \param[out] dispY Pointer to where the y displacement for the vertex that
 *    belongs to this thread should be stored.
 */
__device__ void fa2UpdateDisplacement(unsigned int gid,
    unsigned int numvertices, float speed, float forceX, float forceY,
    float* dispX, float* dispY);

/*!
 * Updates the location of each vertex.
 *
 * \param[in] gid The global ID of this thread.
 * \param[in] numvertices The total number of vertices in the graph.
 * \param[out] vxLocs Array containing the x location of every vertex.
 * \param[out] vyLocs Array containing the y location of every vertex.
 * \param[in] xdisp The x displacement value for the vertex that belongs to this
 *    thread.
 * \param[in] ydisp The y displacement value for the vertex that belongs to this
 *    thread.
 */
__device__ void fa2UpdateLocation(unsigned int gid, unsigned int numvertices,
    float* vxLocs, float* vyLocs, float xdisp, float ydisp);

/*!
 * Computes the out degree for each vertex.
 *
 * \param[in] numvertices The total number of vertices in the graph.
 * \param[in] numedges The total number of edges in the graph.
 * \param[in] edgeSources Array holding the vertex index for each edge-source.
 * \param[out] deg Array holding the out degree for each vertex.
 */
__global__ void fa2ComputeDegrees(unsigned int numvertices,
    unsigned int numedges, unsigned int* edgeSources, unsigned int* deg);

/*!
 * CUDA Kernel that computes the graph swing and graph traction values.
 *
 * \param[in] numvertices The total number of vertices in the graph.
 * \param[in] swg Array holding the swing value for each vertex in the graph.
 * \param[in] tra Array holding the traction value for each vertex in the
 *    graph.
 * \param[in] numNeighbours Array holding the out degree value for each vertex
 *    in the graph.
 * \param[out] graphSwing Pointer to where the graph swing value should be
 *    stored.
 * \param[out] graphTract Pointer to where the graph traction value should be
 *    stored.
 */
__global__ void fa2GraphSwingTract(unsigned int numvertices, float* swg,
    float* tra, unsigned int* numNeighbours, float* graphSwing,
    float* graphTract);

__device__ void fa2Gravity(unsigned int gid, unsigned int numvertices,
    float* vxLocs, float* vyLocs, float* forceX, float* forceY,
    unsigned int* deg)
{
  if (gid < numvertices)
  {
    float vx = vxLocs[gid];
    float vy = vyLocs[gid];
    float vlen = vectorGetLength(vx, vy);
    vectorInverse(&vx, &vy);
    vectorMultiply(&vx, &vy, K_G * (deg[gid] + 1) / vlen);
    if (gid == 0)
      DEBUG_PRINT("g:%f\n", vx);
    vectorAdd(forceX, forceY, vx, vy);
  }
}

__device__ void fa2Repulsion(unsigned int gid, unsigned int numvertices,
    float* vxLocs, float* vyLocs, float* forceX, float* forceY,
    unsigned int* deg)
{
  if (gid < numvertices)
  {
    for (size_t j = 0; j < numvertices; j++)
    {
      if (gid == j)
        continue;
      float vx1 = vxLocs[gid];
      float vy1 = vyLocs[gid];
      float vx2 = vxLocs[j];
      float vy2 = vyLocs[j];

      vectorSubtract(&vx1, &vy1, vx2, vy2);
      float dist = vectorGetLength(vx1, vy1);

      if (dist > 0)
      {
        vectorNormalize(&vx1, &vy1);
        vectorMultiply(&vx1, &vy1,
            K_R * (((deg[gid] + 1) * (deg[j] + 1)) / dist));
        // vectorMultiply(&vx1, &vy1, 0.5);

        vectorAdd(forceX, forceY, vx1, vy1);
      }
    }
  }
}

__device__ void fa2Attraction(unsigned int gid, unsigned int numvertices,
    float* vxLocs, float* vyLocs, unsigned int* numedges,
    unsigned int* edgeTargets, unsigned int maxedges, float* forceX,
    float* forceY)
{
  if (gid < numvertices)
  {
    if (gid == 0)
      DEBUG_PRINT("numedges:%u\n", numedges[gid]);
    float vx1 = vxLocs[gid];
    float vy1 = vyLocs[gid];
    // Each thread goes through its array of edges.
    for (size_t i = 0; i < maxedges; i++)
    {
      unsigned int index = gid + (numvertices * i);
      unsigned int target = edgeTargets[index];
      if (target == UINT_MAX)
        continue;
      // Compute attraction force.
      float vx2 = vxLocs[target];
      float vy2 = vyLocs[target];

      vectorSubtract(&vx2, &vy2, vx1, vy1);
      // vectorMultiply(&vx2, &vy2, 0.5);
      vectorAdd(forceX, forceY, vx2, vy2);
      if (gid == 0)
        DEBUG_PRINT("a:%f\t%u\n", vx2, target);
    }
  }
}

// Updates the swing for each vertex, as described in the Force Atlas 2 paper.
__device__ void fa2UpdateSwing(unsigned int gid, unsigned int numvertices,
    float forceX, float forceY, float* oldForceX, float* oldForceY, float* swg)
{
  if (gid < numvertices)
  {
    float fx = oldForceX[gid];
    float fy = oldForceY[gid];
    vectorSubtract(&fx, &fy, forceX, forceY);
    float vlen = vectorGetLength(fx, fy);
    swg[gid] = vlen;
  }
}

// Updates the traction for each vertex, as described in the Force Atlas 2
// paper.
__device__ void fa2UpdateTract(unsigned int gid, unsigned int numvertices,
    float forceX, float forceY, float* oldForceX, float* oldForceY, float* tra)
{
  if (gid < numvertices)
  {
    float fx = oldForceX[gid];
    float fy = oldForceY[gid];
    vectorAdd(&fx, &fy, forceX, forceY);
    float vlen = vectorGetLength(fx, fy);
    tra[gid] = vlen / 2;
  }
}

/*!
 * Updates the swing value for the graph itself.
 *
 * \param[in] numvertices The total number of vertices in the graph.
 * \param[in] swg Array holding the swing values of each vertex in the graph.
 * \param[in] deg Array holding the out degree values of each vertex in the
 *    graph.
 * \param[out] gswing Pointer to where the graph swing value should be stored.
 */
__device__ void fa2UpdateSwingGraph(unsigned int numvertices, float* swg,
    unsigned int* deg, float* gswing)
{
  __shared__ float scratch[BLOCK_SIZE * 2];

  // Setup local data to perform reduction.
  unsigned int tx = threadIdx.x;
  unsigned int base = tx + (blockIdx.x * BLOCK_SIZE * 2);
  unsigned int stride = BLOCK_SIZE;

  if (base < numvertices)
  {
    scratch[tx] = (deg[base] + 1) * swg[base];
  }
  else
    scratch[tx] = 0;

  if (base + stride < numvertices)
  {
    scratch[tx + stride] = (deg[base + stride] + 1) * swg[base + stride];
  }
  else
    scratch[tx + stride] = 0;

  // Do block-local reduction.
  while (stride > 0)
  {
    __syncthreads();
    if (tx < stride)
    {
      scratch[tx] += scratch[tx + stride];
    }

    stride >>= 1;
  }

  // Do atomic add per block to obtain final value.
  __syncthreads();
  if (tx == 0)
    atomicAdd(gswing, scratch[tx]);
}

/*!
 * Updates the graph traction value.
 *
 * \param[in] numvertices The total number of vertices in the graph.
 * \param[in] tra Array holding the traction values of individual vertices.
 * \param[in] deg Array holding the number of outgoing edges for each vertex.
 * \param[out] gtract Pointer to where the graph traction should be stored.
 */
__device__ void fa2UpdateTractGraph(unsigned int numvertices, float* tra,
    unsigned int* deg, float* gtract)
{
  __shared__ float scratch[BLOCK_SIZE * 2];

  // Setup local data to perform reduction.
  unsigned int tx = threadIdx.x;
  unsigned int base = tx + (blockIdx.x * BLOCK_SIZE * 2);
  unsigned int stride = BLOCK_SIZE;

  if (base < numvertices)
  {
    scratch[tx] = (deg[base] + 1) * tra[base];
  }
  else
    scratch[tx] = 0;

  if (base + stride < numvertices)
  {
    scratch[tx + stride] = (deg[base + stride] + 1) * tra[base + stride];
  }
  else
    scratch[tx + stride] = 0;

  // Do block-local reduction.
  while (stride > 0)
  {
    __syncthreads();
    if (tx < stride)
    {
      scratch[tx] += scratch[tx + stride];
    }

    stride >>= 1;
  }

  // Do atomic add per block to obtain final value.
  __syncthreads();
  if (tx == 0)
    atomicAdd(gtract, scratch[tx]);
}

__device__ void fa2UpdateSpeedGraph(float gswing, float gtract, float* gspeed)
{
  float oldSpeed = *gspeed;

  if (gswing == 0)
  {
    gswing = FLOAT_EPSILON;
  }

  *gspeed = TAU * (gtract / gswing);

  if (oldSpeed > 0 && *gspeed > 1.5 * oldSpeed)
  {
    *gspeed = 1.5 * oldSpeed;
  }

}

__device__ void fa2UpdateSpeed(unsigned int gid, unsigned int numvertices,
    float* speed, float* swg, float forceX, float forceY, float gs)
{
  if (gid < numvertices)
  {
    float vSwg = swg[gid];
    if (vSwg <= 0)
      vSwg = EPSILON;
    float vForceLen = vectorGetLength(forceX, forceY);
    if (vForceLen <= 0)
      vForceLen = EPSILON;

    *speed = K_S * gs / (1 + (gs * sqrt(vSwg)));
  }
}

// Save current forces as the previous forces for the next tick.
__device__ void fa2SaveOldForces(unsigned int gid, unsigned int numvertices,
    float forceX, float forceY, float* oldForceX, float* oldForceY)
{
  if (gid < numvertices)
  {
    oldForceX[gid] = forceX;
    oldForceY[gid] = forceY;
  }
}

__device__ void fa2UpdateDisplacement(unsigned int gid,
    unsigned int numvertices, float speed, float forceX, float forceY,
    float* dispX, float* dispY)
{
  if (gid < numvertices)
  {
    *dispX = forceX;
    *dispY = forceY;
    vectorMultiply(dispX, dispY, speed);
  }
}

__device__ void fa2UpdateLocation(unsigned int gid, unsigned int numvertices,
    float* vxLocs, float* vyLocs, float xdisp, float ydisp)
{
  if (gid < numvertices)
  {
    vxLocs[gid] += xdisp;
    vyLocs[gid] += ydisp;
  }
}

__global__ void fa2ComputeDegrees(unsigned int numvertices,
    unsigned int numedges, unsigned int* edgeSources, unsigned int* deg)
{
  unsigned int gid = threadIdx.x + (blockIdx.x * BLOCK_SIZE);
  if (gid < numvertices)
  {
    unsigned int degree = 0;
    for (size_t i = 0; i < numedges; i++)
    {
      if (edgeSources[i] == gid)
      {
        degree++;
      }
    }
    deg[gid] = degree;
  }
}

__global__ void fa2GraphSwingTract(unsigned int numvertices, float* swg,
    float* tra, unsigned int* numNeighbours, float* graphSwing,
    float* graphTract)
{
  // Update swing of Graph.
  fa2UpdateSwingGraph(numvertices, swg, numNeighbours, graphSwing);

  // Update traction of Graph.
  fa2UpdateTractGraph(numvertices, tra, numNeighbours, graphTract);
}

/*!
 * The kernel that computes the forces on all vertices.
 *
 * \param[in] vxLocs Array holding the x locations for each vertex.
 * \param[in] vyLocs Array holding the y locations for each vertex.
 * \param[in] numvertices The total number of vertices.
 * \param[in] edgeTargets Array representing the edges.
 * \param[in] numedges Array that holds the number of edges for each vertex.
 * \param[in] maxedges the maximum number of edges per vertex.
 * \param[out] tra Array that will be used to store the vertex traction values.
 * \param[out] swg Array that will be used to store the vertex swing values.
 * \param[out] forceX Array that will be used to store the x forces on each vertex.
 * \param[out] forceY Array that will be used to store the y forces on each vertex.
 * \param[in] oldForceX Array that holds the x forces for each vertex from the previous iteration.
 * \param[in] oldForceY Array that holds the y forces for each vertex from the previous iteration.
 */
__global__ void fa2kernel(float* vxLocs, float* vyLocs,
    unsigned int numvertices, unsigned int* edgeTargets, unsigned int* numedges,
    unsigned int maxedges, float* tra, float* swg, float* forceX, float* forceY,
    float* oldForceX, float* oldForceY)
{
  unsigned int gid = threadIdx.x + (blockIdx.x * BLOCK_SIZE);

  if (gid < numvertices)
  {
    forceX[gid] = 0;
    forceY[gid] = 0;
    
    if (gid == 0)
      DEBUG_PRINT("Computing gravity\n");
    // Gravity force
    fa2Gravity(gid, numvertices, vxLocs, vyLocs, &forceX[gid], &forceY[gid],
        numedges);
    if (gid == 0)
      DEBUG_PRINT("Computing repulsion\n");
    // Repulsion between vertices
    fa2Repulsion(gid, numvertices, vxLocs, vyLocs, &forceX[gid], &forceY[gid],
        numedges);
    if (gid == 0)
      DEBUG_PRINT("Computing attraction\n");
    // Attraction on edges
    fa2Attraction(gid, numvertices, vxLocs, vyLocs, numedges, edgeTargets,
        maxedges, &forceX[gid], &forceY[gid]);

    if (gid == 0)
      DEBUG_PRINT("Computing swing\n");
    // Calculate speed of vertices.
    // Update swing of vertices.
    fa2UpdateSwing(gid, numvertices, forceX[gid], forceY[gid], oldForceX,
        oldForceY, swg);

    if (gid == 0)
      DEBUG_PRINT("Computing traction\n");
    // Update traction of vertices.
    fa2UpdateTract(gid, numvertices, forceX[gid], forceY[gid], oldForceX,
        oldForceY, tra);
  }
}

/*!
 * Moves the vertices to their new location after force computation is complete.
 *
 * \param[in,out] vxLocs The x-locations of the vertices. This will be updated with new positions.
 * \param[in,out] vyLocs The y-locations of the vertices. This will be updated with new positions.
 * \param[in] numvertices The total number of vertices.
 * \param[in] tra An array holding the traction values for each vertex.
 * \param[in] swg An array holding the swing values for each vertex.
 * \param[in] forceX An array holding the x forces on each vertex.
 * \param[in] forceY An array holding the y forces on each vertex.
 * \param[in,out] oldForceX An array holding the x force on each vertex of the previous iteration.
 * These values will be overwritten with the current forces.
 * \param[in,out] oldForceY An array holding the y force on each vertex of the previous iteration.
 * These values will be overwritten with the current forces.
 * \param[in] graphSwing the swing value of the graph.
 * \param[in] graphTract the traction value of the graph.
 * \param[in] graphSpeed the speed value of the graph.
 */
__global__ void fa2MoveVertices(float* vxLocs, float* vyLocs,
    unsigned int numvertices, float* tra, float* swg, float* forceX,
    float* forceY, float* oldForceX, float* oldForceY, float* graphSwing,
    float* graphTract, float* graphSpeed)
{
  unsigned int gid = threadIdx.x + (blockIdx.x * BLOCK_SIZE);

  if (gid < numvertices)
  {
    float speed = 0;
    float dispX = 0;
    float dispY = 0;

    // Update speed of Graph.
    fa2UpdateSpeedGraph(*graphSwing, *graphTract, graphSpeed);

    // Update speed of vertices.
    fa2UpdateSpeed(gid, numvertices, &speed, swg, forceX[gid], forceY[gid],
        *graphSpeed);

    // Update displacement of vertices.
    fa2UpdateDisplacement(gid, numvertices, speed, forceX[gid], forceY[gid],
        &dispX, &dispY);

    // Update vertex locations based on speed.
    fa2UpdateLocation(gid, numvertices, vxLocs, vyLocs, dispX, dispY);

    // Set current forces as old forces in vertex data.
    fa2SaveOldForces(gid, numvertices, forceX[gid], forceY[gid], oldForceX,
        oldForceY);
  }
}

/*!
 * Allocated general memory on the device that will be used when running
 * force atlas 2.
 *
 * \param[in,out] data A valid struct where the pointers need to be saved.
 * \param[in] numvertices the number of vertices in the graph.
 */
void fa2PrepareGeneralMemory(ForceAtlas2Data* data, unsigned int numvertices)
{
  // Allocate data for vertices, edges, and fa2 data.
  hipMalloc(&data->tra, numvertices * sizeof(float));
  hipMalloc(&data->swg, numvertices * sizeof(float));
  hipMalloc(&data->forceX, numvertices * sizeof(float));
  hipMalloc(&data->forceY, numvertices * sizeof(float));
  hipMalloc(&data->oldForceX, numvertices * sizeof(float));
  hipMalloc(&data->oldForceY, numvertices * sizeof(float));
  hipMalloc(&data->graphSwing, sizeof(float));
  hipMalloc(&data->graphTract, sizeof(float));
  hipMalloc(&data->graphSpeed, sizeof(float));

  hipMemset(data->tra, 0, numvertices * sizeof(float));
  hipMemset(data->swg, 0, numvertices * sizeof(float));
  hipMemset(data->forceX, 0, numvertices * sizeof(float));
  hipMemset(data->forceY, 0, numvertices * sizeof(float));
  hipMemset(data->oldForceX, 0, numvertices * sizeof(float));
  hipMemset(data->oldForceY, 0, numvertices * sizeof(float));
  hipMemset(data->graphSwing, 0, sizeof(float));
  hipMemset(data->graphTract, 0, sizeof(float));
  hipMemset(data->graphSpeed, 0, sizeof(float));
}

/*!
 * Prepares all memory to run force atlas 2 on the device.
 *
 * \param[in,out] data A valid data object where all pointers should be saved.
 * \param[in] edges The edges that need to be copied to the device.
 * \param[in] numvertices The total number of vertices.
 * \param[in] stream The cuda stream to use while preparing the data.
 */
void fa2PrepareMemory(ForceAtlas2Data* data,
    unsigned int numvertices)
{
  fa2PrepareGeneralMemory(data, numvertices);
}

/*!
 * Cleans the general memory that is required for force atlas 2 on the device.
 * This includes vertex data. This excludes edge data.
 *
 * \param[in] data The struct that holds the data pointers.
 */
void fa2CleanGeneralMemory(ForceAtlas2Data* data)
{
  hipFree(data->tra);
  hipFree(data->swg);
  hipFree(data->forceX);
  hipFree(data->forceY);
  hipFree(data->oldForceX);
  hipFree(data->oldForceY);
  hipFree(data->graphSwing);
  hipFree(data->graphTract);
  hipFree(data->graphSpeed);
}

/*!
 * Cleans the memory on the device that is required for running force atlas 2.
 *
 * \param[in] data The data that needs to be cleaned.
 * \param[in] numvertices The total number of vertices.
 */
void fa2CleanMemory(ForceAtlas2Data* data, unsigned int numvertices)
{
  fa2CleanGeneralMemory(data);
}

void fa2RunOnGraph(Graph* g, unsigned int iterations)
{
  CudaTimer timerIteration, timer;

  // Allocate data for fa2 data.
  ForceAtlas2Data data;
  fa2PrepareMemory(&data, g->vertices->numvertices);

  float* vxLocs = g->vertices->vertexXLocs;
  float* vyLocs = g->vertices->vertexYLocs;
  unsigned int* numEdges = g->edges->numedges;
  unsigned int* edgeTargets = g->edges->edgeTargets;

  unsigned int numblocks = ceil(g->vertices->numvertices / (float) BLOCK_SIZE);
  unsigned int numblocks_reduction = ceil(numblocks / 2.0);

  hipGetLastError();
  hipError_t code = hipGetLastError();
  if (code != hipSuccess)
  {
    printf("Error calculating node degrees.\n%s\n", hipGetErrorString(code));
    exit(EXIT_FAILURE);
  }

  for (size_t i = 0; i < iterations; i++)
  {
    // Run fa2 spring embedding kernel.
    startCudaTimer(&timerIteration);

    // Compute graph speed, vertex forces, speed and displacement.
    startCudaTimer(&timer);
    fa2kernel<<<numblocks, BLOCK_SIZE>>>(vxLocs, vyLocs,
        g->vertices->numvertices, edgeTargets, numEdges,
        g->edges->maxedges, data.tra, data.swg, data.forceX, data.forceY,
        data.oldForceX, data.oldForceY);
    stopCudaTimer(&timer);
    printf("time: all forces and moving vertices.\n");
    printCudaTimer(&timer);
    resetCudaTimer(&timer);
    code = hipGetLastError();
    if (code != hipSuccess)
    {
      printf("Error in kernel 2.\n%s\n", hipGetErrorString(code));
      exit(EXIT_FAILURE);
    }

    stopCudaTimer(&timerIteration);
    printf("time: iteration.\n");
    printCudaTimer(&timerIteration);
    resetCudaTimer(&timerIteration);

    hipMemset(data.graphSwing, 0, sizeof(float));
    hipMemset(data.graphTract, 0, sizeof(float));

    // Run reductions on vertex swing and traction.
    startCudaTimer(&timer);
    fa2GraphSwingTract<<<numblocks_reduction, BLOCK_SIZE>>>(
        g->vertices->numvertices, data.swg, data.tra, numEdges,
        data.graphSwing, data.graphTract);
    stopCudaTimer(&timer);
    printf("time: graph swing and traction.\n");
    printCudaTimer(&timer);
    resetCudaTimer(&timer);

    code = hipGetLastError();
    if (code != hipSuccess)
    {
      printf("Error in kernel 1.\n%s\n", hipGetErrorString(code));
      exit(EXIT_FAILURE);
    }

    fa2MoveVertices<<<numblocks, BLOCK_SIZE>>>(vxLocs, vyLocs,
        g->vertices->numvertices, data.tra, data.swg, data.forceX, data.forceY,
        data.oldForceX, data.oldForceY, data.graphSwing, data.graphTract,
        data.graphSpeed);
  }

  fa2CleanMemory(&data, g->vertices->numvertices);
}

