#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <float.h>
#include "force-atlas-2.h"
#include "math.h"
#include "timer.h"
#include "vector.h"

#define BLOCK_SIZE 64

// Gravity force
__device__ void fa2Gravity(unsigned int gid, unsigned int numvertices,
    float* vxLocs, float* vyLocs, float* forceX, float* forceY,
    unsigned int* deg);
// Repulsion between vertices
__device__ void fa2Repulsion(unsigned int gid, unsigned int numvertices,
    float* vxLocs, float* vyLocs, float* forceX, float* forceY,
    unsigned int* deg);
// Attraction on edges
__device__ void fa2Attraction(unsigned int gid, unsigned int numvertices,
    float* vxLocs, float* vyLocs, int numedges, unsigned int* edgeSources,
    unsigned int* edgeTargets, float* forceX, float* forceY);

__device__ void fa2UpdateSwing(unsigned int gid, unsigned int numvertices,
    float forceX, float forceY, float* oldForceX, float* oldForceY,
    float* swg);
__device__ void fa2UpdateTract(unsigned int gid, unsigned int numvertices,
    float forceX, float forceY, float* oldForceX, float* oldForceY,
    float* tra);
__device__ void fa2UpdateSwingGraph(unsigned int gid, unsigned int numvertices,
    float* swg, int* deg, float* gswing);
__device__ void fa2UpdateTractGraph(unsigned int gid, unsigned int numvertices,
    float* tra, int* deg, float* gtract);
__device__ void fa2UpdateSpeedGraph(float gswing, float gtract, float* gspeed);
__device__ void fa2UpdateSpeed(unsigned int gid, unsigned int numvertices,
    float* speed, float* swg, float forceX, float forceY, float gs);
__device__ void fa2SaveOldForces(unsigned int gid, unsigned int numvertices,
    float forceX, float forceY, float* oldForceX, float* oldForceY);
__device__ void fa2UpdateDisplacement(unsigned int gid,
    unsigned int numvertices, float speed, float forceX, float forceY,
    float* dispX, float* dispY);
__device__ void fa2UpdateLocation(unsigned int gid, unsigned int numvertices,
    float* vxLocs, float* vyLocs, float xdisp, float ydisp);

__global__ void fa2ComputeDegrees(unsigned int numvertices,
    unsigned int numedges, unsigned int* edgeSources, unsigned int* deg);
__global__ void fa2GraphSwingTract(unsigned int numvertices,
    float* swg, float* tra, unsigned int* numNeighbours,
    float* graphSwing, float* graphTract);

__device__ void fa2Gravity(unsigned int gid, unsigned int numvertices,
    float* vxLocs, float* vyLocs, float* forceX, float* forceY,
    unsigned int* deg)
{
  if (gid < numvertices)
  {
    float vx = vxLocs[gid];
    float vy = vyLocs[gid];
    float vlen = vectorGetLength(vx, vy);
    vectorInverse(&vx, &vy);
    vectorMultiply(&vx, &vy, K_G * (deg[gid] + 1) / vlen);
    vectorAdd(forceX, forceY, vx, vy);
  }
}

__device__ void fa2Repulsion(unsigned int gid, unsigned int numvertices,
    float* vxLocs, float* vyLocs, float* forceX, float* forceY,
    unsigned int* deg)
{
  if (gid < numvertices)
  {
    for (size_t j = 0; j < numvertices; j++)
    {
      if (gid == j) continue;
      float vx1 = vxLocs[gid];
      float vy1 = vyLocs[gid];
      float vx2 = vxLocs[j];
      float vy2 = vyLocs[j];

      vectorSubtract(&vx1, &vy1, vx2, vy2);
      float dist = vectorGetLength(vx1, vy1);

      if (dist > 0)
      {
        vectorNormalize(&vx1, &vy1);
        vectorMultiply(&vx1, &vy1, K_R * (((deg[gid] + 1) * (deg[j] + 1))
              / dist));
        // vectorMultiply(&vx1, &vy1, 0.5);

        vectorAdd(forceX, forceY, vx1, vy1);
      }
    }
  }
}

__device__ void fa2Attraction(unsigned int gid, unsigned int numvertices,
    float* vxLocs, float* vyLocs, unsigned int numedges,
    unsigned int* edgeSources, unsigned int* edgeTargets, float* forceX,
    float* forceY)
{
  if (gid < numvertices)
  {
    float vx1 = vxLocs[gid];
    float vy1 = vyLocs[gid];
    // Complete scan on edge array.
    for (size_t i = 0; i < numedges; i++)
    {
      unsigned int source = edgeSources[i];
      unsigned int target = edgeSources[i];
      // Edge source is this vertex.
      if (source == gid)
      {
        // Compute attraction force.
        float vx2 = vxLocs[target];
        float vy2 = vyLocs[target];

        vectorSubtract(&vx2, &vy2, vx1, vy1);
        // vectorMultiply(&vx2, &vy2, 0.5);
        vectorAdd(forceX, forceY, vx2, vy2);
      }
    }
  }
}

// Updates the swing for each vertex, as described in the Force Atlas 2 paper.
__device__ void fa2UpdateSwing(unsigned int gid, unsigned int numvertices,
    float forceX, float forceY, float* oldForceX, float* oldForceY,
    float* swg)
{
  if (gid < numvertices)
  {
    float fx = oldForceX[gid];
    float fy = oldForceY[gid];
    vectorSubtract(&fx, &fy, forceX, forceY);
    float vlen = vectorGetLength(fx, fy);
    swg[gid] = vlen;
  }
}

// Updates the traction for each vertex, as described in the Force Atlas 2
// paper.
__device__ void fa2UpdateTract(unsigned int gid, unsigned int numvertices,
    float forceX, float forceY, float* oldForceX, float* oldForceY,
    float* tra)
{
  if (gid < numvertices)
  {
    float fx = oldForceX[gid];
    float fy = oldForceY[gid];
    vectorAdd(&fx, &fy, forceX, forceY);
    float vlen = vectorGetLength(fx, fy);
    tra[gid] = vlen / 2;
  }
}

// Calculate the current swing of the graph.
__device__ void fa2UpdateSwingGraph(unsigned int gid, unsigned int numvertices,
    float* swg, unsigned int* deg, float* gswing)
{
  __shared__ float scratch[BLOCK_SIZE * 2];

  // Setup local data to perform reduction.
  unsigned int tx = threadIdx.x;
  unsigned int base = tx + (blockIdx.x * BLOCK_SIZE * 2);
  unsigned int stride = BLOCK_SIZE;

  if (base < numvertices)
    scratch[tx] = (deg[base] + 1) * swg[base];
  else
    scratch[tx] = 0;

  if (base + stride < numvertices)
    scratch[tx + stride] = (deg[base + stride] + 1) * swg[base + stride];
  else
    scratch[tx + stride] = 0;

  // Do block-local reduction.
  while (stride > 0)
  {
    __syncthreads();
    if (tx < stride)
    {
      scratch[tx] += scratch[tx + stride];
    }

    stride >>= 1;
  }

  // Do atomic add per block to obtain final value.
  __syncthreads();
  if (tx == 0)
    atomicAdd(gswing, scratch[tx]);
}

// Calculate the current traction of the graph.
__device__ void fa2UpdateTractGraph(unsigned int gid, unsigned int numvertices,
    float* tra, unsigned int* deg, float* gtract)
{
  __shared__ float scratch[BLOCK_SIZE * 2];

  // Setup local data to perform reduction.
  unsigned int tx = threadIdx.x;
  unsigned int base = tx + (blockIdx.x * BLOCK_SIZE * 2);
  unsigned int stride = BLOCK_SIZE;

  if (base < numvertices)
    scratch[tx] = (deg[base] + 1) * tra[base];
  else
    scratch[tx] = 0;

  if (base + stride < numvertices)
    scratch[tx + stride] = (deg[base + stride] + 1) * tra[base + stride];
  else
    scratch[tx + stride] = 0;

  // Do block-local reduction.
  while (stride > 0)
  {
    __syncthreads();
    if (tx < stride)
    {
      scratch[tx] += scratch[tx + stride];
    }

    stride >>= 1;
  }

  // Do atomic add per block to obtain final value.
  __syncthreads();
  if (tx == 0)
    atomicAdd(gtract, scratch[tx]);
}

__device__ void fa2UpdateSpeedGraph(float gswing, float gtract, float* gspeed)
{
  float oldSpeed = *gspeed;

  if (gswing == 0)
    gswing = FLOAT_EPSILON;

  *gspeed = TAU * (gtract / gswing);

  //if (*gspeed <= 0)
  //  *gspeed = EPSILON;
  // Do not allow more then 50% speed increase.
  if (oldSpeed > 0 && *gspeed > 1.5 * oldSpeed)
    *gspeed = 1.5 * oldSpeed;
}

__device__ void fa2UpdateSpeed(unsigned int gid, unsigned int numvertices,
    float* speed, float* swg, float forceX, float forceY, float gs)
{
  if (gid < numvertices)
  {
    float vSwg = swg[gid];
    if (vSwg <= 0)
      vSwg = EPSILON;
    float vForceLen = vectorGetLength(forceX, forceY);
    if (vForceLen <= 0)
      vForceLen = EPSILON;

    *speed = K_S * gs / (1 + (gs * sqrt(vSwg)));
    //*speed = fmin(*speed, (float)
    //    K_SMAX / vForceLen);
  }
}

// Save current forces as the previous forces for the next tick.
__device__ void fa2SaveOldForces(unsigned int gid, unsigned int numvertices,
    float forceX, float forceY, float* oldForceX, float* oldForceY)
{
  if (gid < numvertices)
  {
    oldForceX[gid] = forceX;
    oldForceY[gid] = forceY;
  }
}

__device__ void fa2UpdateDisplacement(unsigned int gid,
    unsigned int numvertices, float speed, float forceX, float forceY,
    float* dispX, float* dispY)
{
  if (gid < numvertices)
  {
    *dispX = forceX;
    *dispY = forceY;
    vectorMultiply(dispX, dispY, speed);
    // FIXME remove this print
    printf("speed:  %f.\nforces:  %f,%f.\n", speed, forceX, forceY);
  }
}

__device__ void fa2UpdateLocation(unsigned int gid, unsigned int numvertices,
    float* vxLocs, float* vyLocs, float xdisp, float ydisp)
{
  if (gid < numvertices)
  {
    vxLocs[gid] += xdisp;
    vyLocs[gid] += ydisp;
  }
}

__global__ void fa2ComputeDegrees(unsigned int numvertices,
    unsigned int numedges, unsigned int* edgeSources, unsigned int* deg)
{
  unsigned int gid = threadIdx.x + (blockIdx.x * BLOCK_SIZE);
  if (gid < numvertices)
  {
    deg[gid] = 0;
    for (size_t i = 0; i < numedges; i++)
    {
      if (edgeSources[i] == gid)
      {
        deg[gid]++;
      }
    }
  }
}

__global__ void fa2GraphSwingTract(unsigned int numvertices,
    float* swg, float* tra, unsigned int* numNeighbours,
    float* graphSwing, float* graphTract)
{
  unsigned int gid = threadIdx.x + (blockIdx.x * BLOCK_SIZE);

  // Update swing of Graph.
  fa2UpdateSwingGraph(gid, numvertices, swg, numNeighbours, graphSwing);

  // Update traction of Graph.
  fa2UpdateTractGraph(gid, numvertices, tra, numNeighbours, graphTract);
}

__global__ void fa2kernel(
    float* vxLocs, float* vyLocs,
    unsigned int numvertices,
    unsigned int* edgeSources,
    unsigned int* edgeTargets,
    unsigned int numedges,
    unsigned int* numNeighbours,
    float* tra, float* swg, 
    float* oldForceX, float* oldForceY,
    float* graphSwing,
    float* graphTract,
    float* oldGraphSpeed)
{
  unsigned int gid = threadIdx.x + (blockIdx.x * BLOCK_SIZE);

  if (gid < numvertices)
  {
    float graphSpeed = *oldGraphSpeed;

    float forceX = 0.0;
    float forceY = 0.0;

    float dispX = 0.0;
    float dispY = 0.0;

    float speed = 0.0;

    // Update speed of Graph.
    fa2UpdateSpeedGraph(*graphSwing, *graphTract, &graphSpeed);

    if (gid == 0)
    {
      *oldGraphSpeed = graphSpeed;
      *graphSwing = 0.0;
      *graphTract = 0.0;
    }

    // Gravity force
    fa2Gravity(gid, numvertices, vxLocs, vyLocs, &forceX, &forceY, numNeighbours);
    // Repulsion between vertices
    fa2Repulsion(gid, numvertices, vxLocs, vyLocs, &forceX, &forceY, numNeighbours);
    // Attraction on edges
    fa2Attraction(gid, numvertices, vxLocs, vyLocs, numedges, edgeSources,
        edgeTargets, &forceX, &forceY);

    // Calculate speed of vertices.
    // Update swing of vertices.
    fa2UpdateSwing(gid, numvertices, forceX, forceY, oldForceX, oldForceY, swg);

    // Update traction of vertices.
    fa2UpdateTract(gid, numvertices, forceX, forceY, oldForceX, oldForceY, tra);

    // Update speed of vertices.
    fa2UpdateSpeed(gid, numvertices, &speed, swg, forceX, forceY, graphSpeed);

    // Update displacement of vertices.
    fa2UpdateDisplacement(gid, numvertices, speed, forceX, forceY, &dispX, &dispY);

    // Set current forces as old forces in vertex data.
    fa2SaveOldForces(gid, numvertices, forceX, forceY, oldForceX, oldForceY);

    // Update vertex locations based on speed.
    // TODO Add a barrier here to make sure no vertex location is update before
    // all vertices have calculated their repulsion and attraction forces.
    fa2UpdateLocation(gid, numvertices, vxLocs, vyLocs, dispX, dispY);
  }
}

void fa2RunOnGraph(Graph* g, unsigned int iterations)
{
  // Make variables for vertices, edges and fa2 data.
  unsigned int* numNeighbours = NULL;
  float* tra = NULL;
  float* swg = NULL;
  float* oldForceX = NULL;
  float* oldForceY = NULL;
  float* graphSwing = NULL;
  float* graphTract = NULL;
  float* graphSpeed = NULL;

  float* vxLocs = NULL;
  float* vyLocs = NULL;
  unsigned int* edgeSources = NULL;
  unsigned int* edgeTargets = NULL;

  Timer timerMem1, timerMem2;
  //hipEvent_t start, stop;
  //float time;

  // Allocate data for vertices, edges, and fa2 data.
  hipMalloc(&numNeighbours, g->numvertices * sizeof(int));
  hipMalloc(&tra, g->numvertices * sizeof(float));
  hipMalloc(&swg, g->numvertices * sizeof(float));
  hipMalloc(&oldForceX, g->numvertices * sizeof(float));
  hipMalloc(&oldForceY, g->numvertices * sizeof(float));
  hipMalloc(&graphSwing, sizeof(float));
  hipMalloc(&graphTract, sizeof(float));
  hipMalloc(&graphSpeed, sizeof(float));

  hipMemset(numNeighbours, 0, g->numvertices * sizeof(int));
  hipMemset(tra, 0, g->numvertices * sizeof(float));
  hipMemset(swg, 0, g->numvertices * sizeof(float));
  hipMemset(oldForceX, 0, g->numvertices * sizeof(float));
  hipMemset(oldForceY, 0, g->numvertices * sizeof(float));
  hipMemset(graphSwing, 0, sizeof(float));
  hipMemset(graphTract, 0, sizeof(float));
  hipMemset(graphSpeed, 0, sizeof(float));

  hipMalloc(&vxLocs, g->numvertices * sizeof(float));
  hipMalloc(&vyLocs, g->numvertices * sizeof(float));
  hipMalloc(&edgeSources, g->numedges * sizeof(unsigned int));
  hipMalloc(&edgeTargets, g->numedges * sizeof(unsigned int));

  startTimer(&timerMem1);

  // Copy vertices and edges to device.
  hipMemcpy((void*) vxLocs, g->vertexXLocs, g->numvertices * sizeof(float),
      hipMemcpyHostToDevice);
  hipMemcpy((void*) vyLocs, g->vertexYLocs, g->numvertices * sizeof(float),
      hipMemcpyHostToDevice);
  hipMemcpy((void*) edgeSources, g->edgeSources,
      g->numedges * sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy((void*) edgeTargets, g->edgeTargets,
      g->numedges * sizeof(unsigned int), hipMemcpyHostToDevice);

  stopTimer(&timerMem1);

  unsigned int numblocks = ceil(g->numvertices / (float) BLOCK_SIZE);
  unsigned int numblocks_reduction = ceil(numblocks / 2.0);

  // Compute vertex degrees using current edges.
  fa2ComputeDegrees<<<numblocks, BLOCK_SIZE>>>(g->numvertices, g->numedges,
      edgeSources, numNeighbours);

  for (size_t i = 0; i < iterations; i++)
  {
    // Run fa2 spring embedding kernel.

    //hipEventCreate(&start);
    //hipEventRecord(start, 0);

    *graphSwing = *graphTract = 0; 

    // Run reductions on vertex swing and traction.
    fa2GraphSwingTract<<<numblocks_reduction, BLOCK_SIZE>>>(
        g->numvertices,
        swg, tra, numNeighbours,
        graphSwing, graphTract);

    hipError_t code = hipGetLastError();
    if (code != hipSuccess)
    {
      printf("Error in kernel 1.\n%s\n", hipGetErrorString(code));
      exit(EXIT_FAILURE);
    }

    //hipEventCreate(&stop);
    //hipEventRecord(stop, 0);
    //hipEventSynchronize(stop);
    //hipEventElapsedTime(&time, start, stop);
    //hipEventDestroy(start);
    //hipEventDestroy(stop);
    printf("time: graph swing and traction.\n");
    printf("(ms)  %f.\n", time);

    //hipEventCreate(&start);
    //hipEventRecord(start, 0);

    printf("Launching %i blocks of size %i. Which is %i threads.\n", numblocks,
      BLOCK_SIZE, numblocks * BLOCK_SIZE);
    printf("We have %i vertices and %i edges.\n", g->numvertices, g->numedges);

    // Compute graph speed, vertex forces, speed and displacement.
    fa2kernel<<<numblocks, BLOCK_SIZE>>>(
        vxLocs,
        vyLocs,
        g->numvertices,
        edgeSources,
        edgeTargets,
        g->numedges,
        numNeighbours,
        tra,
        swg,
        oldForceX,
        oldForceY,
        graphSwing,
        graphTract,
        graphSpeed);
    code = hipGetLastError();
    if (code != hipSuccess)
    {
      printf("Error in kernel 2.\n%s\n", hipGetErrorString(code));
      exit(EXIT_FAILURE);
    }

    //hipEventCreate(&stop);
    //hipEventRecord(stop, 0);
    //hipEventSynchronize(stop);
    //hipEventElapsedTime(&time, start, stop);
    //hipEventDestroy(start);
    //hipEventDestroy(stop);
    printf("time: all forces and moving vertices.\n");
    printf("(ms)  %f.\n", time);
  }

  startTimer(&timerMem2);

  // Update graph with new vertex positions.
  hipMemcpy((void*) g->vertexXLocs, vxLocs, g->numvertices * sizeof(float),
      hipMemcpyDeviceToHost);
  hipMemcpy((void*) g->vertexYLocs, vyLocs, g->numvertices * sizeof(float),
      hipMemcpyDeviceToHost);
  hipMemcpy((void*) g->edgeSources, edgeSources,
      g->numedges * sizeof(unsigned int), hipMemcpyDeviceToHost);
  hipMemcpy((void*) g->edgeTargets, edgeTargets,
      g->numedges * sizeof(unsigned int), hipMemcpyDeviceToHost);

  stopTimer(&timerMem2);
  printf("time: copying data host to device.\n");
  printTimer(&timerMem1);
  printf("time: copying data device to host.\n");
  printTimer(&timerMem2);

  hipFree(numNeighbours);
  hipFree(tra);
  hipFree(swg);
  hipFree(oldForceX);
  hipFree(oldForceY);
  hipFree(graphSwing);
  hipFree(graphTract);
  hipFree(graphSpeed);
}

