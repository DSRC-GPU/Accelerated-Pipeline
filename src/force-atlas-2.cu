#include "hip/hip_runtime.h"
/*!
 * \file force-atlas-2.cu
 * A parallel implementation of the Force Atlas 2 spring embedding algorithm.
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <float.h>
#include "force-atlas-2.h"
#include "math.h"
#include "cuda-timer.h"
#include "vector.h"

/*!
 * The number of thread in each thread block.
 */
#define BLOCK_SIZE 64

/*!
 * Updates the current force on each vertex with the current gravity.
 *
 * \param[in] gid The global ID of this thread.
 * \param[in] numvertices The total number of vertices in the graph.
 * \param[in] vxLocs Array that holds the x location of all vertices.
 * \param[in] vyLocs Array that holds the y location of all vertices.
 * \param[out] forceX Pointer to the x force on the vertex that belongs to this
 *    thread.
 * \param[out] forceY Pointer to the y force on the vertex that belongs to this
 *    thread.
 * \param[in] deg Array holding the out degree values for each vertex.
 */
__device__ void fa2Gravity(unsigned int gid, unsigned int numvertices,
    float* vxLocs, float* vyLocs, float* forceX, float* forceY,
    unsigned int* deg);

/*!
 * Updates the current force on each vertex with the current repulsion.
 *
 * \param[in] gid The global ID of this thread.
 * \param[in] numvertices The total number of vertices in the graph.
 * \param[in] vxLocs Array that holds the x location of all vertices.
 * \param[in] vyLocs Array that holds the y location of all vertices.
 * \param[out] forceX Pointer to the x force on the vertex that belongs to this
 *    thread.
 * \param[out] forceY Pointer to the y force on the vertex that belongs to this
 *    thread.
 * \param[in] deg Array holding the out degree values for each vertex.
 */
__device__ void fa2Repulsion(unsigned int gid, unsigned int numvertices,
    float* vxLocs, float* vyLocs, float* forceX, float* forceY,
    unsigned int* deg);

/*!
 * Updates the current force on each vertex with the current attraction.
 *
 * \param[in] gid The global ID of this thread.
 * \param[in] numvertices The total number of vertices in the graph.
 * \param[in] vxLocs Array that holds the x location of all vertices.
 * \param[in] vyLocs Array that holds the y location of all vertices.
 * \param[in] numedges The total number of edges in the graph.
 * \param[in] edgeSources Array holding the vertex ID for each edge source in
 *    order.
 * \param[in] edgeTargets Array holding the vertex ID for each edge target in
 *    order.
 * \param[out] forceX Pointer to the x force on the vertex that belongs to this
 *    thread.
 * \param[out] forceY Pointer to the y force on the vertex that belongs to this
 *    thread.
 */
__device__ void fa2Attraction(unsigned int gid, unsigned int numvertices,
    float* vxLocs, float* vyLocs, unsigned int numedges, unsigned int* edgeSources,
    unsigned int* edgeTargets, float* forceX, float* forceY);

/*!
 * Updates the swing value for each vertex in the graph.
 *
 * \param[in] gid The global ID of this thread.
 * \param[in] numvertices The total number of vertices in the graph.
 * \param[in] forceX The x force value on the vertex that belongs to this
 *    thread.
 * \param[in] forceY The y force value on the vertex that belongs to this
 *    thread.
 * \param[in] oldForceX Array holding the x forces of each vertex from the
 *    previous iteration.
 * \param[in] oldForceY Array holding the y forces of each vertex from the
 *    previous iteration.
 * \param[out] swg Array where the swing values for each vertex should be
 *    stored.
 */
__device__ void fa2UpdateSwing(unsigned int gid, unsigned int numvertices,
    float forceX, float forceY, float* oldForceX, float* oldForceY,
    float* swg);

/*!
 * Updates the traction value for each vertex in the graph.
 *
 * \param[in] gid The global ID of this thread.
 * \param[in] numvertices The total number of vertices in the graph.
 * \param[in] forceX The x force value on the vertex that belongs to this
 *    thread.
 * \param[in] forceY The y force value on the vertex that belongs to this
 *    thread.
 * \param[in] oldForceX Array holding the x forces of each vertex from the
 *    previous iteration.
 * \param[in] oldForceY Array holding the y forces of each vertex from the
 *    previous iteration.
 * \param[out] swg Array where the traction values for each vertex should be
 *    stored.
 */
__device__ void fa2UpdateTract(unsigned int gid, unsigned int numvertices,
    float forceX, float forceY, float* oldForceX, float* oldForceY,
    float* tra);

/*!
 * Updates the swing value for the graph itself.
 *
 * \param[in] numvertices The total number of vertices in the graph.
 * \param[in] swg Array holding the swing values of each vertex in the graph.
 * \param[in] deg Array holding the out degree values of each vertex in the
 *    graph.
 * \param[out] gswing Pointer to where the graph swing value should be stored.
 */
__device__ void fa2UpdateSwingGraph(unsigned int numvertices,
    float* swg, int* deg, float* gswing);

/*!
 * Updates the traction value for the graph itself.
 *
 * \param[in] numvertices The total number of vertices in the graph.
 * \param[in] tra Array holding the traction values of each vertex in the graph.
 * \param[in] deg Array holding the out degree values of each vertex in the
 *    graph.
 * \param[out] gtract Pointer to where the graph traction value should be stored.
 */
__device__ void fa2UpdateTractGraph(unsigned int numvertices,
    float* tra, int* deg, float* gtract);

/*!
 * Updates the speed value for the graph itself.
 *
 * \param[in] gswing The swing value of the graph.
 * \param[in] gtract The traction value of the graph.
 * \param[out] gspeed Pointer where the graph speed should be stored.
 */
__device__ void fa2UpdateSpeedGraph(float gswing, float gtract, float* gspeed);

/*!
 * Updates the speed value for each vertex in the graph.
 *
 * \param[in] gid The global ID of this thread.
 * \param[in] numvertices The total number of vertices in the graph.
 * \param[out] speed Pointer to where the speed value of the vertex that belongs
 *    to this thread should be stored.
 * \param[in] forceX The x force on the vertex that belongs to this thread.
 * \param[in] forceY The y force on the vertex that belongs to this thread.
 * \param[in] gs The graph speed value.
 */
__device__ void fa2UpdateSpeed(unsigned int gid, unsigned int numvertices,
    float* speed, float* swg, float forceX, float forceY, float gs);

/*!
 * Copies the forces of this iteration to another array. Overwrites the values
 * in the destination array.
 *
 * \param[in] gid The global ID of this thread.
 * \param[in] numvertices The total number of vertices in the graph.
 * \param[in] forceX The x force on the vertex that belongs to this thread.
 * \param[in] forceY The y force on the vertex that belongs to this thread.
 * \param[out] oldForceX Array that will be used to store the x forces of this
 *    iteration.
 * \param[out] oldForceY Array that will be used to store the y forces of this
 *    iteration.
 */
__device__ void fa2SaveOldForces(unsigned int gid, unsigned int numvertices,
    float forceX, float forceY, float* oldForceX, float* oldForceY);

/*!
 * Updates the vertex displacement array.
 *
 * \param[in] gid The global ID of this thread.
 * \param[in] numvertices The total number of vertices in the graph.
 * \param[in] speed The speed value for the vertex that belongs to this thread.
 * \param[in] forceX The x force value for the vertex that belongs to this
 *    thread.
 * \param[in] forceY The y force value for the vertex that belongs to this
 *    thread.
 * \param[out] dispX Pointer to where the x displacement for the vertex that
 *    belongs to this thread should be stored.
 * \param[out] dispY Pointer to where the y displacement for the vertex that
 *    belongs to this thread should be stored.
 */
__device__ void fa2UpdateDisplacement(unsigned int gid,
    unsigned int numvertices, float speed, float forceX, float forceY,
    float* dispX, float* dispY);

/*!
 * Updates the location of each vertex.
 *
 * \param[in] gid The global ID of this thread.
 * \param[in] numvertices The total number of vertices in the graph.
 * \param[out] vxLocs Array containing the x location of every vertex.
 * \param[out] vyLocs Array containing the y location of every vertex.
 * \param[in] xdisp The x displacement value for the vertex that belongs to this
 *    thread.
 * \param[in] ydisp The y displacement value for the vertex that belongs to this
 *    thread.
 */
__device__ void fa2UpdateLocation(unsigned int gid, unsigned int numvertices,
    float* vxLocs, float* vyLocs, float xdisp, float ydisp);

/*!
 * Computes the out degree for each vertex.
 *
 * \param[in] numvertices The total number of vertices in the graph.
 * \param[in] numedges The total number of edges in the graph.
 * \param[in] edgeSources Array holding the vertex index for each edge-source.
 * \param[out] deg Array holding the out degree for each vertex.
 */
__global__ void fa2ComputeDegrees(unsigned int numvertices,
    unsigned int numedges, unsigned int* edgeSources, unsigned int* deg);

/*!
 * CUDA Kernel that computes the graph swing and graph traction values.
 *
 * \param[in] numvertices The total number of vertices in the graph.
 * \param[in] swg Array holding the swing value for each vertex in the graph.
 * \param[in] tra Array holding the traction value for each vertex in the
 *    graph.
 * \param[in] numNeighbours Array holding the out degree value for each vertex
 *    in the graph.
 * \param[out] graphSwing Pointer to where the graph swing value should be
 *    stored.
 * \param[out] graphTract Pointer to where the graph traction value should be
 *    stored.
 */
__global__ void fa2GraphSwingTract(unsigned int numvertices,
    float* swg, float* tra, unsigned int* numNeighbours,
    float* graphSwing, float* graphTract);

__device__ void fa2Gravity(unsigned int gid, unsigned int numvertices,
    float* vxLocs, float* vyLocs, float* forceX, float* forceY,
    unsigned int* deg)
{
  if (gid < numvertices)
  {
    float vx = vxLocs[gid];
    float vy = vyLocs[gid];
    float vlen = vectorGetLength(vx, vy);
    vectorInverse(&vx, &vy);
    vectorMultiply(&vx, &vy, K_G * (deg[gid] + 1) / vlen);
    vectorAdd(forceX, forceY, vx, vy);
  }
}

__device__ void fa2Repulsion(unsigned int gid, unsigned int numvertices,
    float* vxLocs, float* vyLocs, float* forceX, float* forceY,
    unsigned int* deg)
{
  if (gid < numvertices)
  {
    for (size_t j = 0; j < numvertices; j++)
    {
      if (gid == j) continue;
      float vx1 = vxLocs[gid];
      float vy1 = vyLocs[gid];
      float vx2 = vxLocs[j];
      float vy2 = vyLocs[j];

      vectorSubtract(&vx1, &vy1, vx2, vy2);
      float dist = vectorGetLength(vx1, vy1);

      if (dist > 0)
      {
        vectorNormalize(&vx1, &vy1);
        vectorMultiply(&vx1, &vy1, K_R * (((deg[gid] + 1) * (deg[j] + 1))
              / dist));
        // vectorMultiply(&vx1, &vy1, 0.5);

        vectorAdd(forceX, forceY, vx1, vy1);
      }
    }
  }
}

__device__ void fa2Attraction(unsigned int gid, unsigned int numvertices,
    float* vxLocs, float* vyLocs, unsigned int numedges,
    unsigned int* edgeSources, unsigned int* edgeTargets, float* forceX,
    float* forceY)
{
  if (gid < numvertices)
  {
    float vx1 = vxLocs[gid];
    float vy1 = vyLocs[gid];
    // Complete scan on edge array.
    for (size_t i = 0; i < numedges; i++)
    {
      unsigned int source = edgeSources[i];
      unsigned int target = edgeTargets[i];
      // Edge source is this vertex.
      if (source == gid)
      {
        // Compute attraction force.
        float vx2 = vxLocs[target];
        float vy2 = vyLocs[target];

        vectorSubtract(&vx2, &vy2, vx1, vy1);
        // vectorMultiply(&vx2, &vy2, 0.5);
        vectorAdd(forceX, forceY, vx2, vy2);
      }
    }
  }
}

// Updates the swing for each vertex, as described in the Force Atlas 2 paper.
__device__ void fa2UpdateSwing(unsigned int gid, unsigned int numvertices,
    float forceX, float forceY, float* oldForceX, float* oldForceY,
    float* swg)
{
  if (gid < numvertices)
  {
    float fx = oldForceX[gid];
    float fy = oldForceY[gid];
    vectorSubtract(&fx, &fy, forceX, forceY);
    float vlen = vectorGetLength(fx, fy);
    swg[gid] = vlen;
  }
}

// Updates the traction for each vertex, as described in the Force Atlas 2
// paper.
__device__ void fa2UpdateTract(unsigned int gid, unsigned int numvertices,
    float forceX, float forceY, float* oldForceX, float* oldForceY,
    float* tra)
{
  if (gid < numvertices)
  {
    float fx = oldForceX[gid];
    float fy = oldForceY[gid];
    vectorAdd(&fx, &fy, forceX, forceY);
    float vlen = vectorGetLength(fx, fy);
    tra[gid] = vlen / 2;
  }
}

// Calculate the current swing of the graph.
__device__ void fa2UpdateSwingGraph(unsigned int numvertices,
    float* swg, unsigned int* deg, float* gswing)
{
  __shared__ float scratch[BLOCK_SIZE * 2];

  // Setup local data to perform reduction.
  unsigned int tx = threadIdx.x;
  unsigned int base = tx + (blockIdx.x * BLOCK_SIZE * 2);
  unsigned int stride = BLOCK_SIZE;

  if (base < numvertices)
  {
    scratch[tx] = (deg[base] + 1) * swg[base];
  }
  else
    scratch[tx] = 0;

  if (base + stride < numvertices)
  {
    scratch[tx + stride] = (deg[base + stride] + 1) * swg[base + stride];
  }
  else
    scratch[tx + stride] = 0;

  // Do block-local reduction.
  while (stride > 0)
  {
    __syncthreads();
    if (tx < stride)
    {
      scratch[tx] += scratch[tx + stride];
    }

    stride >>= 1;
  }

  // Do atomic add per block to obtain final value.
  __syncthreads();
  if (tx == 0)
    atomicAdd(gswing, scratch[tx]);
}

// Calculate the current traction of the graph.
__device__ void fa2UpdateTractGraph(unsigned int numvertices,
    float* tra, unsigned int* deg, float* gtract)
{
  __shared__ float scratch[BLOCK_SIZE * 2];

  // Setup local data to perform reduction.
  unsigned int tx = threadIdx.x;
  unsigned int base = tx + (blockIdx.x * BLOCK_SIZE * 2);
  unsigned int stride = BLOCK_SIZE;

  if (base < numvertices)
  {
    scratch[tx] = (deg[base] + 1) * tra[base];
  }
  else
    scratch[tx] = 0;

  if (base + stride < numvertices)
  {
    scratch[tx + stride] = (deg[base + stride] + 1) * tra[base + stride];
  }
  else
    scratch[tx + stride] = 0;

  // Do block-local reduction.
  while (stride > 0)
  {
    __syncthreads();
    if (tx < stride)
    {
      scratch[tx] += scratch[tx + stride];
    }

    stride >>= 1;
  }

  // Do atomic add per block to obtain final value.
  __syncthreads();
  if (tx == 0)
    atomicAdd(gtract, scratch[tx]);
}

__device__ void fa2UpdateSpeedGraph(float gswing, float gtract, float* gspeed)
{
  float oldSpeed = *gspeed;

  if (gswing == 0)
  {
    gswing = FLOAT_EPSILON;
  }

  *gspeed = TAU * (gtract / gswing);

  if (oldSpeed > 0 && *gspeed > 1.5 * oldSpeed)
  {
    *gspeed = 1.5 * oldSpeed;
  }

}

__device__ void fa2UpdateSpeed(unsigned int gid, unsigned int numvertices,
    float* speed, float* swg, float forceX, float forceY, float gs)
{
  if (gid < numvertices)
  {
    float vSwg = swg[gid];
    if (vSwg <= 0)
      vSwg = EPSILON;
    float vForceLen = vectorGetLength(forceX, forceY);
    if (vForceLen <= 0)
      vForceLen = EPSILON;

    *speed = K_S * gs / (1 + (gs * sqrt(vSwg)));
  }
}

// Save current forces as the previous forces for the next tick.
__device__ void fa2SaveOldForces(unsigned int gid, unsigned int numvertices,
    float forceX, float forceY, float* oldForceX, float* oldForceY)
{
  if (gid < numvertices)
  {
    oldForceX[gid] = forceX;
    oldForceY[gid] = forceY;
  }
}

__device__ void fa2UpdateDisplacement(unsigned int gid,
    unsigned int numvertices, float speed, float forceX, float forceY,
    float* dispX, float* dispY)
{
  if (gid < numvertices)
  {
    *dispX = forceX;
    *dispY = forceY;
    vectorMultiply(dispX, dispY, speed);
  }
}

__device__ void fa2UpdateLocation(unsigned int gid, unsigned int numvertices,
    float* vxLocs, float* vyLocs, float xdisp, float ydisp)
{
  if (gid < numvertices)
  {
    vxLocs[gid] += xdisp;
    vyLocs[gid] += ydisp;
  }
}

__global__ void fa2ComputeDegrees(unsigned int numvertices,
    unsigned int numedges, unsigned int* edgeSources, unsigned int* deg)
{
  unsigned int gid = threadIdx.x + (blockIdx.x * BLOCK_SIZE);
  if (gid < numvertices)
  {
    unsigned int degree = 0;
    for (size_t i = 0; i < numedges; i++)
    {
      if (edgeSources[i] == gid)
      {
        degree++;
      }
    }
    deg[gid] = degree;
  }
}

__global__ void fa2GraphSwingTract(unsigned int numvertices,
    float* swg, float* tra, unsigned int* numNeighbours,
    float* graphSwing, float* graphTract)
{
  // Update swing of Graph.
  fa2UpdateSwingGraph(numvertices, swg, numNeighbours, graphSwing);

  // Update traction of Graph.
  fa2UpdateTractGraph(numvertices, tra, numNeighbours, graphTract);
}

__global__ void fa2kernel(
    float* vxLocs, float* vyLocs,
    unsigned int numvertices,
    unsigned int* edgeSources,
    unsigned int* edgeTargets,
    unsigned int numedges,
    unsigned int* numNeighbours,
    float* tra, float* swg, 
    float* forceX, float* forceY,
    float* oldForceX, float* oldForceY,
    float* graphSwing,
    float* graphTract,
    float* oldGraphSpeed)
{
  unsigned int gid = threadIdx.x + (blockIdx.x * BLOCK_SIZE);

  if (gid < numvertices)
  {
    forceX[gid] = 0;
    forceY[gid] = 0;

    // Gravity force
    fa2Gravity(gid, numvertices, vxLocs, vyLocs, &forceX[gid], &forceY[gid], numNeighbours);
    // Repulsion between vertices
    fa2Repulsion(gid, numvertices, vxLocs, vyLocs, &forceX[gid], &forceY[gid], numNeighbours);
    // Attraction on edges
    fa2Attraction(gid, numvertices, vxLocs, vyLocs, numedges, edgeSources,
        edgeTargets, &forceX[gid], &forceY[gid]);

    // Calculate speed of vertices.
    // Update swing of vertices.
    fa2UpdateSwing(gid, numvertices, forceX[gid], forceY[gid], oldForceX, oldForceY, swg);

    // Update traction of vertices.
    fa2UpdateTract(gid, numvertices, forceX[gid], forceY[gid], oldForceX, oldForceY, tra);
  }
}

__global__ void fa2MoveVertices(
    float* vxLocs, float* vyLocs,
    unsigned int numvertices,
    float* tra, float* swg,
    float* forceX, float* forceY,
    float* oldForceX, float* oldForceY,
    float* graphSwing,
    float* graphTract,
    float* graphSpeed)
{
  unsigned int gid = threadIdx.x + (blockIdx.x * BLOCK_SIZE);

  if (gid < numvertices)
  {
    float speed = 0;
    float dispX = 0;
    float dispY = 0;

    // Update speed of Graph.
    fa2UpdateSpeedGraph(*graphSwing, *graphTract, graphSpeed);

    // Update speed of vertices.
    fa2UpdateSpeed(gid, numvertices, &speed, swg, forceX[gid], forceY[gid], *graphSpeed);

    // Update displacement of vertices.
    fa2UpdateDisplacement(gid, numvertices, speed, forceX[gid], forceY[gid], &dispX, &dispY);

    // Update vertex locations based on speed.
    fa2UpdateLocation(gid, numvertices, vxLocs, vyLocs, dispX, dispY);

    // Set current forces as old forces in vertex data.
    fa2SaveOldForces(gid, numvertices, forceX[gid], forceY[gid], oldForceX, oldForceY);
  }
}

void fa2RunOnGraph(Graph* g, unsigned int iterations)
{
  // Make variables for vertices, edges and fa2 data.
  unsigned int* numNeighbours = NULL;
  float* tra = NULL;
  float* swg = NULL;
  float* forceX = NULL;
  float* forceY = NULL;
  float* oldForceX = NULL;
  float* oldForceY = NULL;
  float* graphSwing = NULL;
  float* graphTract = NULL;
  float* graphSpeed = NULL;

  float* vxLocs = NULL;
  float* vyLocs = NULL;
  unsigned int* edgeSources = NULL;
  unsigned int* edgeTargets = NULL;

  CudaTimer timerMem1, timerMem2, timerIteration, timer;

  // Allocate data for vertices, edges, and fa2 data.
  hipMalloc(&numNeighbours, g->vertices->numvertices * sizeof(int));
  hipMalloc(&tra, g->vertices->numvertices * sizeof(float));
  hipMalloc(&swg, g->vertices->numvertices * sizeof(float));
  hipMalloc(&forceX, g->vertices->numvertices * sizeof(float));
  hipMalloc(&forceY, g->vertices->numvertices * sizeof(float));
  hipMalloc(&oldForceX, g->vertices->numvertices * sizeof(float));
  hipMalloc(&oldForceY, g->vertices->numvertices * sizeof(float));
  hipMalloc(&graphSwing, sizeof(float));
  hipMalloc(&graphTract, sizeof(float));
  hipMalloc(&graphSpeed, sizeof(float));

  hipMemset(numNeighbours, 0, g->vertices->numvertices * sizeof(int));
  hipMemset(tra, 0, g->vertices->numvertices * sizeof(float));
  hipMemset(swg, 0, g->vertices->numvertices * sizeof(float));
  hipMemset(forceX, 0, g->vertices->numvertices * sizeof(float));
  hipMemset(forceY, 0, g->vertices->numvertices * sizeof(float));
  hipMemset(oldForceX, 0, g->vertices->numvertices * sizeof(float));
  hipMemset(oldForceY, 0, g->vertices->numvertices * sizeof(float));
  hipMemset(graphSwing, 0, sizeof(float));
  hipMemset(graphTract, 0, sizeof(float));
  hipMemset(graphSpeed, 0, sizeof(float));

  hipMalloc(&vxLocs, g->vertices->numvertices * sizeof(float));
  hipMalloc(&vyLocs, g->vertices->numvertices * sizeof(float));
  hipMalloc(&edgeSources, g->edges->numedges * sizeof(unsigned int));
  hipMalloc(&edgeTargets, g->edges->numedges * sizeof(unsigned int));

  startCudaTimer(&timerMem1);

  // Copy vertices and edges to device.
  hipMemcpy((void*) vxLocs, g->vertices->vertexXLocs, g->vertices->numvertices * sizeof(float),
      hipMemcpyHostToDevice);
  hipMemcpy((void*) vyLocs, g->vertices->vertexYLocs, g->vertices->numvertices * sizeof(float),
      hipMemcpyHostToDevice);
  hipMemcpy((void*) edgeSources, g->edges->edgeSources,
      g->edges->numedges * sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy((void*) edgeTargets, g->edges->edgeTargets,
      g->edges->numedges * sizeof(unsigned int), hipMemcpyHostToDevice);

  stopCudaTimer(&timerMem1);

  unsigned int numblocks = ceil(g->vertices->numvertices / (float) BLOCK_SIZE);
  unsigned int numblocks_reduction = ceil(numblocks / 2.0);

  hipGetLastError();

  // Compute vertex degrees using current edges.
  fa2ComputeDegrees<<<numblocks, BLOCK_SIZE>>>(g->vertices->numvertices, g->edges->numedges,
      edgeSources, numNeighbours);

  hipError_t code = hipGetLastError();
  if (code != hipSuccess)
  {
    printf("Error calculating node degrees.\n%s\n", hipGetErrorString(code));
    exit(EXIT_FAILURE);
  }

  for (size_t i = 0; i < iterations; i++)
  {
    // Run fa2 spring embedding kernel.
    startCudaTimer(&timerIteration);

    // Compute graph speed, vertex forces, speed and displacement.
    startCudaTimer(&timer);
    fa2kernel<<<numblocks, BLOCK_SIZE>>>(
        vxLocs,
        vyLocs,
        g->vertices->numvertices,
        edgeSources,
        edgeTargets,
        g->edges->numedges,
        numNeighbours,
        tra,
        swg,
        forceX,
        forceY,
        oldForceX,
        oldForceY,
        graphSwing,
        graphTract,
        graphSpeed);
    stopCudaTimer(&timer);
    printf("time: all forces and moving vertices.\n");
    printCudaTimer(&timer);
    resetCudaTimer(&timer);
    code = hipGetLastError();
    if (code != hipSuccess)
    {
      printf("Error in kernel 2.\n%s\n", hipGetErrorString(code));
      exit(EXIT_FAILURE);
    }

    stopCudaTimer(&timerIteration);
    printf("time: iteration.\n");
    printCudaTimer(&timerIteration);
    resetCudaTimer(&timerIteration);

    hipMemset(graphSwing, 0, sizeof(float));
    hipMemset(graphTract, 0, sizeof(float));

    // Run reductions on vertex swing and traction.
    startCudaTimer(&timer);
    fa2GraphSwingTract<<<numblocks_reduction, BLOCK_SIZE>>>(
        g->vertices->numvertices,
        swg, tra, numNeighbours,
        graphSwing, graphTract);
    stopCudaTimer(&timer);
    printf("time: graph swing and traction.\n");
    printCudaTimer(&timer);
    resetCudaTimer(&timer);

    code = hipGetLastError();
    if (code != hipSuccess)
    {
      printf("Error in kernel 1.\n%s\n", hipGetErrorString(code));
      exit(EXIT_FAILURE);
    }

    fa2MoveVertices<<<numblocks, BLOCK_SIZE>>>(
        vxLocs,
        vyLocs,
        g->vertices->numvertices,
        tra,
        swg,
        forceX,
        forceY,
        oldForceX,
        oldForceY,
        graphSwing,
        graphTract,
        graphSpeed);
  }

  startCudaTimer(&timerMem2);

  // Update graph with new vertex positions.
  hipMemcpy((void*) g->vertices->vertexXLocs, vxLocs, g->vertices->numvertices * sizeof(float),
      hipMemcpyDeviceToHost);
  hipMemcpy((void*) g->vertices->vertexYLocs, vyLocs, g->vertices->numvertices * sizeof(float),
      hipMemcpyDeviceToHost);
  hipMemcpy((void*) g->edges->edgeSources, edgeSources,
      g->edges->numedges * sizeof(unsigned int), hipMemcpyDeviceToHost);
  hipMemcpy((void*) g->edges->edgeTargets, edgeTargets,
      g->edges->numedges * sizeof(unsigned int), hipMemcpyDeviceToHost);

  stopCudaTimer(&timerMem2);
  printf("time: copying data from host to device.\n");
  printCudaTimer(&timerMem1);
  printf("time: copying data from device to host.\n");
  printCudaTimer(&timerMem2);
  resetCudaTimer(&timerMem1);
  resetCudaTimer(&timerMem2);

  hipFree(numNeighbours);
  hipFree(tra);
  hipFree(swg);
  hipFree(oldForceX);
  hipFree(oldForceY);
  hipFree(graphSwing);
  hipFree(graphTract);
  hipFree(graphSpeed);
}

