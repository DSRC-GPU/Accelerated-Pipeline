/*
 * \file cuda-stream.c
 */

#include "cuda-stream.h"

hipStream_t* createCudaStreams(unsigned int num)
{
  hipStream_t* streams = (hipStream_t*) calloc(num, sizeof(hipStream_t));
  for (unsigned int i = 0; i < num; i++)
  {
    // TODO Do not ignore potential error.
    hipError_t error = hipStreamCreate(&streams[i]);
  }
  return streams;
}

void cleanCudaStreams(hipStream_t* streams, unsigned int num)
{
  for (unsigned int i = 0; i < num; i++)
  {
    hipStreamDestroy(streams[i]);
  }
}
