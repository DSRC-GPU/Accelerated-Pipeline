/*
 * \file speecvector.c
 */

#include "speedvector.h"
#include "util.h"

// Note: this could be done faster if each function would only call one kernel.

void speedVectorInit(float** averageSpeedX, float** averageSpeedY,
    float* vxLocs, float* vyLocs, unsigned int numvertices)
{
  hipMalloc(averageSpeedX, numvertices);
  hipMalloc(averageSpeedY, numvertices);
  hipMemset(*averageSpeedX, 0, sizeof(float) * numvertices);
  hipMemset(*averageSpeedY, 0, sizeof(float) * numvertices);
  utilVectorAdd(*averageSpeedX, vxLocs, numvertices);
  utilVectorMultiply(*averageSpeedX, vxLocs, numvertices);
  utilVectorAdd(*averageSpeedY, vyLocs, numvertices);
  utilVectorMultiply(*averageSpeedY, vyLocs, numvertices);
}

void speedVectorUpdate(float* vxLocs, float* vyLocs, float* averageSpeedX,
    float* averageSpeedY, unsigned int numvertices, hipStream_t* stream)
{
  utilVectorAddInStream(averageSpeedX, vxLocs, numvertices, stream);
  utilVectorAddInStream(averageSpeedY, vyLocs, numvertices, stream);
}

void speedVectorFinish(float* averageSpeedX, float* averageSpeedY,
    unsigned int numiterations, unsigned int numvertices)
{
  utilVectorDevideByScalar(averageSpeedX, numiterations, numvertices);
  utilVectorDevideByScalar(averageSpeedY, numiterations, numvertices);
}

void speedVectorClean(float* averageSpeedX, float* averageSpeedY)
{
  hipFree(averageSpeedX);
  hipFree(averageSpeedY);
}
