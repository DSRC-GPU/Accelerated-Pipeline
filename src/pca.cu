#include "hip/hip_runtime.h"
/*!
 * \file
 */

#include "pca.h"
#include "hipblas.h"
#include "cula.h"
#include "stdio.h"
#include "util.h"

float* pca(float* d_inMatrix, unsigned int inCols, unsigned int inRows,
    float* d_outMatrix)
{
  float* d_Y;
  float* d_PC;
  float* d_Signals;

  hipMalloc(&d_PC, inCols * inCols * sizeof(float));
  utilVectorSetByScalar(d_PC, 0, inCols * inCols);

  // Subtract mean for each dimension.
  pcaUpdateMean(d_inMatrix, inRows, inCols);

  // Calculate matrix Y.
  pcaCalculateYMatrix(d_inMatrix, inRows, inCols, d_Y);

  // Perform SVD on Y.
  pcaSVD(d_Y, inRows, inCols, d_PC);
  // TODO Wrap code below in a function call.

  float* h_inMatrix = (float*) calloc(inCols * inRows, sizeof(float));
  h_inMatrix[0] = 1.0;
  h_inMatrix[7] = 4.0;
  h_inMatrix[9] = 3.0;
  h_inMatrix[16] = 2.0;
  hipMalloc(&d_inMatrix, inCols * inRows * sizeof(float));
  hipMemcpy(d_inMatrix, h_inMatrix, inCols * inRows * sizeof(float),
      hipMemcpyHostToDevice);
  //utilVectorSetByScalar(d_inMatrix, 1, inCols * inRows);

  for (size_t i = 0; i < inRows * inCols; i++)
  {
    printf("%f\n", h_inMatrix[i]);
  }
  printf("&&&\n");

  // Calculate signals.
  pcaCalculateSignals(d_PC, d_inMatrix, inRows, inCols, d_Signals);

  // Return signals
  return d_Signals;
}

void pcaUpdateMean(float* d_inMatrix, unsigned int inRows, unsigned int inCols)
{

}

void pcaCalculateYMatrix(float* d_inMatrix, unsigned int inRows, unsigned int
    inCols, float* d_Y)
{

}

void pcaSVD(float* d_Y, unsigned int inRows, unsigned int inCols, float* d_PC)
{
  char jobu = 'N';
  char jobvt = 'A';

  const float alpha = 1.0f;
  const float beta = 0.0f;

  int min = (inCols < inRows) ? inCols : inRows;
  float* S;
  hipMalloc(&S, min * sizeof(float));
  utilVectorSetByScalar(S, 0, min);
  float* d_U;
  hipMalloc(&d_U, inRows * inRows * sizeof(float));
  utilVectorSetByScalar(d_U, 0, inRows * inRows);

  culaInitialize();
  culaDeviceSgesvd(jobu, jobvt, inRows, inCols, d_Y, inRows, S, d_U,
      inRows, d_PC, inCols);
  culaShutdown();

  // TODO Free some memory.
  hipFree(S);
  hipFree(d_U);
}

void pcaCalculateSignals(float* d_PC, float* d_inMatrix, unsigned int inRows,
    unsigned int inCols, float* d_Signals)
{

}

