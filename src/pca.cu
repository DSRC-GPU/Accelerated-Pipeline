#include "hip/hip_runtime.h"
/*!
 * \file
 */

#include "pca.h"
#include "hipblas.h"
#include "cula.h"
#include <stdio.h>
#include "util.h"

void pca(float* d_inMatrix, unsigned int inRows, unsigned int inCols,
    float* d_outMatrix)
{
  float* d_Y = NULL;
  float* d_PC = NULL;

  hipMalloc(&d_PC, inCols * inCols * sizeof(float));
  utilVectorSetByScalar(d_PC, 0, inCols * inCols);

  hipMalloc(&d_Y, inRows * inCols * sizeof(float));
  utilVectorSetByScalar(d_Y, 0, inRows * inCols);

  // Subtract mean for each dimension.
  pcaUpdateMean(d_inMatrix, inRows, inCols);

  // Calculate matrix Y.
  pcaCalculateYMatrix(d_inMatrix, inRows, inCols, d_Y);

  // Perform SVD on Y.
  pcaSVD(d_Y, inCols, inRows, d_PC);

  DEBUG_PRINT_DEVICE(d_PC, inRows * inRows);
  DEBUG_PRINT_DEVICE(d_inMatrix, inRows * inCols);

  // Calculate signals.
  pcaCalculateSignals(d_PC, d_inMatrix, inRows, inCols, d_outMatrix);
}

void pcaUpdateMean(float* d_inMatrix, unsigned int inRows, unsigned int inCols)
{
  float* d_averageX = NULL;
  float* d_averageY = NULL;
  hipMalloc(&d_averageX, sizeof(float));
  hipMalloc(&d_averageY, sizeof(float));

  // Compute the average X and Y values.
  utilParallelSum(&d_inMatrix[0], inCols, d_averageX);
  utilParallelSum(&d_inMatrix[inCols], inCols, d_averageY);

  float h_averageX, h_averageY;
  hipMemcpy(&h_averageX, d_averageX, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&h_averageY, d_averageY, sizeof(float), hipMemcpyDeviceToHost);
  h_averageX /= inCols;
  h_averageY /= inCols;

  utilVectorAddScalar(&d_inMatrix[0], -1 * h_averageX, inCols);
  utilVectorAddScalar(&d_inMatrix[inCols], -1 * h_averageY, inCols);
}

void pcaCalculateYMatrix(float* d_inMatrix, unsigned int inRows, unsigned int
    inCols, float* d_Y)
{
  // Transpose inMatrix
  // Do not transpose because CULA expects column major ordering.
  //
  // const float alpha = 1;
  // const float beta = 0;
  // hipblasHandle_t handle;
  // hipblasCreate(&handle);
  // hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, inCols, inRows, &alpha,
  //     d_inMatrix, inRows, &beta, NULL, inRows, d_Y, inCols);
  // hipblasDestroy(handle);
  
  // Devide all values by sqrt(N-1)
  float sqrtN = sqrt(inCols - 1);
  hipMemcpy(d_Y, d_inMatrix, inRows * inCols * sizeof(float),
      hipMemcpyDeviceToDevice);
  utilVectorDevideByScalar(d_Y, sqrtN, inRows * inCols); 
}

void pcaSVD(float* d_Y, unsigned int inRows, unsigned int inCols, float* d_PC)
{
  char jobu = 'N';
  char jobvt = 'A';

  int min = (inCols < inRows) ? inCols : inRows;
  float* S;
  hipMalloc(&S, min * sizeof(float));
  utilVectorSetByScalar(S, 0, min);
  float* d_U;
  hipMalloc(&d_U, inRows * inRows * sizeof(float));
  utilVectorSetByScalar(d_U, 0, inRows * inRows);

  culaInitialize();
  culaDeviceSgesvd(jobu, jobvt, inRows, inCols, d_Y, inRows, S, d_U,
      inRows, d_PC, inCols);
  culaShutdown();

  hipFree(S);
  hipFree(d_U);
}

void pcaCalculateSignals(float* d_PC, float* d_inMatrix, unsigned int inRows,
    unsigned int inCols, float* d_Signals)
{
  const float alpha = 1;
  const float beta = 0;
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  float m = inRows;
  float n = inCols;
  float k = inRows;
  float lda = inRows;
  float ldb = inCols;
  float ldc = inRows;

  hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, k, &alpha,
      d_PC, lda, d_inMatrix, ldb, &beta, d_Signals, ldc);

  hipblasDestroy(handle);
}

