#include "hip/hip_runtime.h"
/*!
 * \file
 */

#include "pca.h"
#include "hipblas.h"
#include "cula.h"

void pca(float* d_inMatrix, unsigned int inCols, unsigned int inRows,
    float* d_outMatrix, unsigned int outCols, unsigned int outRows)
{
  char jobu = 'A';
  char jobvt = 'N';

  const float alpha = 1.0f;
  const float beta = 0.0f;

  int min = (inCols < inRows) ? inCols : inRows;
  float* S;
  hipMalloc(&S, min * sizeof(float));
  float* d_U;
  hipMalloc(&d_U, inRows * inRows * sizeof(float));
  float* d_VT;
  hipMalloc(&d_VT, inCols * inCols * sizeof(float));

  culaInitialize();
  culaDeviceSgesvd(jobu, jobvt, inRows, inCols, d_inMatrix, inRows, S, d_U,
      inRows, d_VT, inCols);
  culaShutdown();

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, inRows, inCols, inRows, &alpha,
      d_U, outRows, d_inMatrix, inRows, &beta, d_outMatrix, inRows);
  hipblasDestroy(handle);
}
