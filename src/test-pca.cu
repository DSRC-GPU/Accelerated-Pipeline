
#include "test-pca.h"
#include "pca.h"
#include "stdio.h"

void testPca()
{
  unsigned int m = 2;
  unsigned int n = 10;

  float* h_In;
  float* h_Out;
  h_In = (float*) calloc(m * n, sizeof(float));
  h_Out = (float*) calloc(m * n, sizeof(float));
  for (size_t i = 0; i < m * n; i++)
  {
    h_In[i] = (float) i;
  }
  
  float* d_In;
  float* d_Out;
  hipMalloc(&d_In, m * n * sizeof(float));
  hipMalloc(&d_Out, m * n * sizeof(float));
  hipMemcpy(d_In, h_In, m * n * sizeof(float), hipMemcpyHostToDevice);

  pca(d_In, m, n, d_Out);

  hipMemcpy(h_In, d_In, m * n * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(h_Out, d_Out, m * n * sizeof(float), hipMemcpyDeviceToHost);

  printf("&&&\n");
  for (size_t i = 0; i < m * n; i++)
  {
    printf("%f\n", h_In[i]);
  }
  printf("&&&&\n");
  for (size_t i = 0; i < m * n; i++)
  {
    printf("%f\n", h_Out[i]);
  }
}

//void testCalcYMatrix()
//{
//  unsigned int m = 5;
//  unsigned int n = 2;
//
//  float* h_In;
//  float* h_Out;
//  h_In = (float*) calloc(m * n, sizeof(float));
//  h_Out = (float*) calloc(m * n, sizeof(float));
//  for (size_t i = 0; i < m * n; i++)
//  {
//    h_In[i] = (float) i;
//  }
//  
//  float* d_In;
//  float* d_Out;
//  hipMalloc(&d_In, m * n * sizeof(float));
//  hipMalloc(&d_Out, m * n * sizeof(float));
//  hipMemcpy(d_In, h_In, m * n * sizeof(float), hipMemcpyHostToDevice);
//
//  pcaCalculateYMatrix(d_In, m, n, d_Out);
//
//  hipMemcpy(h_In, d_In, m * n * sizeof(float), hipMemcpyDeviceToHost);
//  hipMemcpy(h_Out, d_Out, m * n * sizeof(float), hipMemcpyDeviceToHost);
//
//  for (size_t i = 0; i < m * n; i++)
//  {
//    printf("%f\n", h_In[i]);
//  }
//  printf("&&&&\n");
//  for (size_t i = 0; i < m * n; i++)
//  {
//    printf("%f\n", h_Out[i]);
//  }
//}

