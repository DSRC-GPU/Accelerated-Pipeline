#include "hip/hip_runtime.h"
/*
 * \file vector-smoothening.c
 */

#include "smoothening.h"
#include "util.h"
#include <stdio.h>


/*!
 * This smoothening function is not completely synchronized because it does not
 * use a global barrier.
 */
__global__ void smootheningRunKernel(float* xvectors,
    unsigned int numvertices, unsigned int* numedges, unsigned int* edges,
    float phi, float* valuesOut)
{
  unsigned int gid = threadIdx.x + (blockIdx.x * BLOCK_SIZE);
  float values;
  if (gid < numvertices)
  {
    // DEBUG_PRINT("%u, %f\n", gid, xvectors[gid]);
    values = phi * xvectors[gid];
    for (size_t i = 0; i < numedges[gid]; i++)
    {
      unsigned int index = edges[gid + (numvertices * i)];
      values += ((1 - phi) * valuesOut[index]) / numedges[gid];
    }
  }
  __syncthreads();
  // if (gid == 0)
  //   DEBUG_PRINT("change: %f\n", xvectors[gid] - values);
  if (gid < numvertices)
    valuesOut[gid] = values;
}

void smootheningPrepareEdges(unsigned int* hostEdges,
    unsigned int* hostNumEdges, unsigned int totaledges,
    unsigned int totalvertices, unsigned int** edges, unsigned int** numedges)
{
  hipMalloc(edges, totaledges * sizeof(unsigned int));
  hipMalloc(numedges, totalvertices * sizeof(unsigned int));
  hipMemcpy(*edges, hostEdges, totaledges * sizeof(unsigned int),
      hipMemcpyHostToDevice);
  hipMemcpy(*numedges, hostNumEdges, totalvertices * sizeof(unsigned int),
      hipMemcpyHostToDevice);
}

void smootheningCleanEdges(unsigned int* edges, unsigned int* numedges)
{
  hipFree(edges);
  hipFree(numedges);
}

void smootheningRun(float* values,
    unsigned int numvertices, unsigned int* numedges, unsigned int* edges,
    unsigned int numiterations, float phi, float* valuesOut)
{
  // Initialize the smoothened values as the input.
  hipMemcpy(valuesOut, values, numvertices * sizeof(float),
      hipMemcpyDeviceToDevice);

  unsigned int numblocks = ceil(numvertices / (float) BLOCK_SIZE);
  for (size_t i = 0; i < numiterations; i++)
  {
    hipGetLastError();
    smootheningRunKernel<<<numblocks, BLOCK_SIZE>>>(values,
        numvertices, numedges, edges, phi, valuesOut);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
      printf("Cuda error: %s\n", hipGetErrorString(err));
      exit (EXIT_FAILURE);
    }
  }

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  utilCudaCheckError(&err, "Error smoothening edges");
}

