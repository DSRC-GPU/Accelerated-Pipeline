#include "timer.h"
#include <stdio.h>

/*!
 * Struct used to compute execution times in a parallel environmennt.
 */
typedef struct ParTimer {
  /*!
   * Struct to save the time when measuring is started.
   */
  hipEvent_t start;
  /*!
   * Struct to save the time when measuring is ended.
   */
  hipEvent_t stop;
} ParTimer;

Timer* timerNew()
{
  Timer* timer = calloc(1, sizeof(Timer));
  timer->internals = calloc(1, sizeof(ParTimer));
  return timer;
}

void timerClean(Timer* timer)
{
  if (timer)
  {
    free(timer->internals);
    free(timer);
  }
}

void startTimer(Timer* timer)
{
  ParTimer* ptimer = (ParTimer*) timer->internals;
  hipEventCreate(&ptimer->start);
  hipEventRecord(ptimer->start, 0);
}

void stopTimer(Timer* timer)
{
  ParTimer* ptimer = (ParTimer*) timer->internals;
  hipEventCreate(&ptimer->stop);
  hipEventRecord(ptimer->stop, 0);
  hipEventSynchronize(ptimer->stop);
}

void resetTimer(Timer* timer)
{
  ParTimer* ptimer = (ParTimer*) timer->internals;
  hipEventDestroy(ptimer->start);
  hipEventDestroy(ptimer->stop);
}

void printTimer(Timer* timer, char* msg)
{
  ParTimer* ptimer = (ParTimer*) timer->internals;
  float time;
  hipEventElapsedTime(&time, ptimer->start, ptimer->stop);
  printf("timer: %s\nkernel time (ms): %f\n", msg, time);
}

