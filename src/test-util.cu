
#include "test-util.h"
#include "util.h"
#include <stdio.h>

void testUtil()
{
  unsigned int numelems = 1000;

  float* h_M = (float*) calloc(numelems, sizeof(float));
  for (size_t i = 0; i < numelems; i++)
  {
    h_M[i] = 1;
  }

  float* d_M = NULL;
  hipMalloc(&d_M, numelems * sizeof(float));
  hipMemcpy(d_M, h_M, numelems * sizeof(float), hipMemcpyHostToDevice);

  float* d_outVal = NULL;
  hipMalloc(&d_outVal, sizeof(float));

  utilParallelSum(d_M, numelems, d_outVal);

  float h_outVal = -1;
  hipMemcpy(&h_outVal, d_outVal, sizeof(float), hipMemcpyDeviceToHost);

  printf("Sum: %f\n", h_outVal);
}

