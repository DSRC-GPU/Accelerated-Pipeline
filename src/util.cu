#include "hip/hip_runtime.h"
/*
 * \file util.cu
 */

#include "util.h"

__global__ void utilVectorAddKernel(float* dst, float* src, unsigned int num)
{
  unsigned int gid = threadIdx.x + BLOCK_SIZE * blockIdx.x;
  if (gid < num)
    dst[gid] += src[gid];
}

__global__ void utilVectorMultiplyKernel(float* dst, float* src,
    unsigned int num)
{
  unsigned int gid = threadIdx.x + BLOCK_SIZE * blockIdx.x;
  if (gid < num)
    dst[gid] *= src[gid];
}

__global__ void utilVectorDevideKernel(float* dst, float* src, unsigned int num)
{
  unsigned int gid = threadIdx.x + BLOCK_SIZE * blockIdx.x;
  if (gid < num)
    dst[gid] /= src[gid];
}

__global__ void utilVectorDevideByScalarKernel(float* dst, float denumerator,
    unsigned int num)
{
  unsigned int gid = threadIdx.x + BLOCK_SIZE * blockIdx.x;
  if (gid < num)
    dst[gid] /= denumerator;
}

void utilVectorAddInStream(float* dst, float* src, unsigned int num,
    hipStream_t* stream)
{
  unsigned int numblocks = ceil(num / (float) BLOCK_SIZE);
  utilVectorAddKernel<<<numblocks, BLOCK_SIZE, 0, *stream>>>(dst, src, num);
}

void utilVectorAdd(float* dst, float* src, unsigned int num)
{
  unsigned int numblocks = ceil(num / (float) BLOCK_SIZE);
  utilVectorAddKernel<<<numblocks, BLOCK_SIZE>>>(dst, src, num);
}

void utilVectorMultiply(float* dst, float* src, unsigned int num)
{
  unsigned int numblocks = ceil(num / (float) BLOCK_SIZE);
  utilVectorMultiplyKernel<<<numblocks, BLOCK_SIZE>>>(dst, src, num);
}

void utilVectorDevide(float* dst, float* src, unsigned int num)
{
  unsigned int numblocks = ceil(num / (float) BLOCK_SIZE);
  utilVectorDevideKernel<<<numblocks, BLOCK_SIZE>>>(dst, src, num);
}

void utilVectorDevideByScalar(float* dst, float denumerator, unsigned int num)
{
  unsigned int numblocks = ceil(num / (float) BLOCK_SIZE);
  utilVectorDevideByScalarKernel<<<numblocks, BLOCK_SIZE>>>(dst, denumerator,
      num);
}
