#include "cuda-timer.h"
#include <stdio.h>

void startCudaTimer(CudaTimer* cTimer)
{
  hipEventCreate(&cTimer->start);
  hipEventRecord(cTimer->start, 0);
}

void stopCudaTimer(CudaTimer* cTimer)
{
  hipEventCreate(&cTimer->stop);
  hipEventRecord(cTimer->stop, 0);
  hipEventSynchronize(cTimer->stop);
}

void resetCudaTimer(CudaTimer* cTimer)
{
  hipEventDestroy(cTimer->start);
  hipEventDestroy(cTimer->stop);
}

void printCudaTimer(CudaTimer* cTimer, char* msg)
{
  float time;
  hipEventElapsedTime(&time, cTimer->start, cTimer->stop);
  printf("timer: %s\nkernel time (ms): %f.\n", msg, time);
}

