#include "cuda-timer.h"
#include <stdio.h>

void startCudaTimer(CudaTimer* cTimer)
{
  hipEventCreate(&cTimer->start);
  hipEventRecord(cTimer->start, 0);
}

void stopCudaTimer(CudaTimer* cTimer)
{
  hipEventCreate(&cTimer->stop);
  hipEventRecord(cTimer->stop, 0);
  hipEventSynchronize(cTimer->stop);
}

void resetCudaTimer(CudaTimer* cTimer)
{
  hipEventDestroy(cTimer->start);
  hipEventDestroy(cTimer->stop);
}

void printCudaTimer(CudaTimer* cTimer)
{
  float time;
  hipEventElapsedTime(&time, cTimer->start, cTimer->stop);
  printf("kernel time (ms): %f.\n", time);
}

