#include "hip/hip_runtime.h"
/*
 * \file vector-smoothening.c
 */

#include "vector-smoothening.h"
#include "util.h"
#include <stdio.h>


/*!
 * This smoothening function is not completely synchronized because it does not
 * use a global barrier.
 */
__global__ void vectorSmootheningRunKernel(float* xvectors, float* yvectors,
    unsigned int numvertices, unsigned int* numedges, unsigned int* edges,
    float phi, float* xvectorsOut, float* yvectorsOut)
{
  unsigned int gid = threadIdx.x + (blockIdx.x * BLOCK_SIZE);
  float newvectorx, newvectory;
  if (gid < numvertices)
  {
    DEBUG_PRINT("%u, %f\n", gid, xvectors[gid]);
    newvectorx = phi * xvectors[gid];
    newvectory = phi * yvectors[gid];
    for (size_t i = 0; i < numedges[gid]; i++)
    {
      unsigned int index = edges[gid + (numvertices * i)];
      newvectorx += ((1 - phi) * xvectorsOut[index]) / numedges[gid];
      newvectory += ((1 - phi) * yvectorsOut[index]) / numedges[gid];
    }
  }
  __syncthreads();
  if (gid == 0)
    DEBUG_PRINT("change: %f\n", xvectors[gid] - newvectorx);
  xvectorsOut[gid] = newvectorx;
  yvectorsOut[gid] = newvectory;
}

void vectorSmootheningPrepareEdges(unsigned int* hostEdges,
    unsigned int* hostNumEdges, unsigned int totaledges,
    unsigned int totalvertices, unsigned int** edges, unsigned int** numedges)
{
  hipMalloc(edges, totaledges * sizeof(unsigned int));
  hipMalloc(numedges, totalvertices * sizeof(unsigned int));
  hipMemcpy(*edges, hostEdges, totaledges * sizeof(unsigned int),
      hipMemcpyHostToDevice);
  hipMemcpy(*numedges, hostNumEdges, totalvertices * sizeof(unsigned int),
      hipMemcpyHostToDevice);
}

void vectorSmootheningPrepareOutput(float** xoutput, float** youtput,
    unsigned int numvertices)
{
  hipMalloc(xoutput, numvertices * sizeof(float));
  hipMalloc(youtput, numvertices * sizeof(float));
}

void vectorSmootheningCleanEdges(unsigned int* edges, unsigned int* numedges)
{
  hipFree(edges);
  hipFree(numedges);
}

void vectorSmootheningRun(float* xvectors, float* yvectors,
    unsigned int numvertices, unsigned int* numedges, unsigned int* edges,
    unsigned int numiterations, float phi, float* xvectorsOut, float* yvectorsOut)
{
  // Copy vectors. These will be the constant vectors used for smoothening. The
  // input array will be used for the smoothened values.
  hipMemcpy(xvectorsOut, xvectors, numvertices * sizeof(float),
      hipMemcpyDeviceToDevice);
  hipMemcpy(yvectorsOut, yvectors, numvertices * sizeof(float),
      hipMemcpyDeviceToDevice);

  unsigned int numblocks = ceil(numvertices / (float) BLOCK_SIZE);
  for (size_t i = 0; i < numiterations; i++)
  {
    hipGetLastError();
    vectorSmootheningRunKernel<<<numblocks, BLOCK_SIZE>>>(xvectors,
        yvectors, numvertices, numedges, edges, phi, xvectorsOut, yvectorsOut);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
      printf("Cuda error: %s\n", hipGetErrorString(err));
      exit (EXIT_FAILURE);
    }
  }
}
